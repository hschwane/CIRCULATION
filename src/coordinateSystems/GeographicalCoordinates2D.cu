#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * GeographicalCoordinates2D.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the GeographicalCoordinates2D class
 *
 * Copyright (c) 2019 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "GeographicalCoordinates2D.h"
//--------------------

// function definitions of the GeographicalCoordinates2D class
//-------------------------------------------------------------------

GeographicalCoordinates2D::GeographicalCoordinates2D(float minLat, float maxLat, int3 numGridCells, float radius)
    : m_radius(radius), m_numGridCells(make_int2(numGridCells)),
        m_min(make_float2(0,minLat)), m_max(make_float2(2* M_PIf32, maxLat)),
        m_totalNumGridCells(numGridCells.x*numGridCells.y), m_size(m_max - m_min),
        m_cellSize( m_size / make_float2(m_numGridCells))
{
}

float3 GeographicalCoordinates2D::getCartesian(const float3& coord) const
{
    float phi = M_PI_2f32 - coord.y;
    float sinPhi = sin(phi);
    return make_float3( m_radius * cos(coord.x) * sinPhi, m_radius * sin(coord.x) * sinPhi, m_radius * cos(phi));
}

float3 GeographicalCoordinates2D::getCoord(const float3& cartesian) const
{
    float r = sqrt( cartesian.x * cartesian.x + cartesian.y * cartesian.y + cartesian.z * cartesian.z );
    float phi = acos(cartesian.z / r);
    return make_float3(  atan2(cartesian.y,cartesian.x), M_PI_2f32 - phi, 0);
}

float3 GeographicalCoordinates2D::getCellCoordinate(int cellId) const
{
    return getCellCoordinate3d({cellId%m_numGridCells.x, cellId/m_numGridCells.x,0});
}

float3 GeographicalCoordinates2D::getCellCoordinate3d(const int3& cellId3d) const
{
    int2 cellId2d = make_int2(cellId3d);
    float2 coord2d = make_float2(cellId2d) * m_cellSize + m_min;
    return make_float3(coord2d);
}

int GeographicalCoordinates2D::getCellId(const float3& coord) const
{
    int3 cellId3d = getCellId3d(coord);
    return cellId3d.y*m_numGridCells.x+ cellId3d.x;
}

int3 GeographicalCoordinates2D::getCellId3d(const float3& coord) const
{
    float2 coord2d =  (make_float2(coord) - m_min) / m_cellSize;
    return make_int3(rintf(coord2d.x),rintf(coord2d.y),0);
}

int GeographicalCoordinates2D::getRightNeighbor(int cellId) const
{
    return cellId+1;
}

int GeographicalCoordinates2D::getLeftNeighbor(int cellId) const
{
    return cellId-1;
}

int GeographicalCoordinates2D::getForwardNeighbor(int cellId) const
{
    return cellId+m_numGridCells.x;
}

int GeographicalCoordinates2D::getBackwardNeighbor(int cellId) const
{
    return cellId-m_numGridCells.x;
}

int GeographicalCoordinates2D::getUpNeighbor(int cellId) const
{
    return -1;
}

int GeographicalCoordinates2D::getDownNeighbor(int cellId) const
{
    return -1;
}

float3 GeographicalCoordinates2D::getMinCoord() const
{
    return make_float3(m_min);
}

float3 GeographicalCoordinates2D::getMaxCoord() const
{
    return make_float3(m_max);
}

int GeographicalCoordinates2D::getNumGridCells() const
{
    return m_totalNumGridCells;
}

int3 GeographicalCoordinates2D::getNumGridCells3d() const
{
    return make_int3(m_numGridCells);
}

float3 GeographicalCoordinates2D::getCellSize() const
{
    return make_float3(m_cellSize);
}

int GeographicalCoordinates2D::getDimension() const
{
    return 2;
}

int GeographicalCoordinates2D::getCartesianDimension() const
{
    return 3;
}

float3 GeographicalCoordinates2D::getAABBMin() const
{
    return make_float3(-m_radius);
}

float3 GeographicalCoordinates2D::getAABBMax() const
{
    return make_float3(m_radius);
}

std::string GeographicalCoordinates2D::getShaderDefine() const
{
    return "GEOGRAPHICAL_COORDINATES_2D";
}

void GeographicalCoordinates2D::setShaderUniforms(mpu::gph::ShaderProgram& shader) const
{
    shader.uniform2f("csInternalData.m_min", glm::vec2(m_min.x,m_min.y));
    shader.uniform2f("csInternalData.m_max", glm::vec2(m_max.x,m_max.y));
    shader.uniform2f("csInternalData.m_size", glm::vec2(m_size.x,m_size.y));
    shader.uniform2f("csInternalData.m_cellSize", glm::vec2(m_cellSize.x,m_cellSize.y));
    shader.uniform2i("csInternalData.m_numGridCells", glm::ivec2(m_numGridCells.x,m_numGridCells.y));
    shader.uniform1i("csInternalData.m_totalNumGridCells", m_totalNumGridCells);
    shader.uniform1f("csInternalData.m_radius", m_radius);
}

