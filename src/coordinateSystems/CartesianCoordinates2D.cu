#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * CartesianCoordinates2D.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the CartesianCoordinates2D class
 *
 * Copyright (c) 2019 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "CartesianCoordinates2D.h"
//--------------------

// function definitions of the CartesianCoordinates2D class
//-------------------------------------------------------------------
CartesianCoordinates2D::CartesianCoordinates2D(float3 min, float3 max, int3 numGridCells)
    : m_min(make_float2(min)), m_max(make_float2(max)),
    m_numGridCells(make_int2(numGridCells)),
    m_totalNumGridCells(numGridCells.x*numGridCells.y),
    m_size(m_max-m_min),
    m_cellSize( m_size / make_float2( (m_numGridCells.x<2) ? 1 : m_numGridCells.x-1, (m_numGridCells.y<2) ? 1 : m_numGridCells.y-1))
{
}

float3 CartesianCoordinates2D::getCartesian(const float3& coord) const
{
    return float3{coord.x,coord.y,0};
}

float3 CartesianCoordinates2D::getCoord(const float3& cartesian) const
{
    return float3{cartesian.x,cartesian.y,0};
}

float3 CartesianCoordinates2D::getUnitVectorX(float3 position) const
{
    return make_float3(1.0f,0.0f,0.0f);
}

float3 CartesianCoordinates2D::getUnitVectorY(float3 position) const
{
    return make_float3(0.0f,1.0f,0.0f);
}

float3 CartesianCoordinates2D::getUnitVectorZ(float3 position) const
{
    return make_float3(0.0f,0.0f,1.0f);
}

float3 CartesianCoordinates2D::getCellCoordinate(int cellId) const
{
    return getCellCoordinate3d({cellId%m_numGridCells.x, cellId/m_numGridCells.x,0});
}

float3 CartesianCoordinates2D::getCellCoordinate3d(const int3& cellId3d) const
{
    int2 cellId2d = make_int2(cellId3d);
    float2 coord2d = make_float2(cellId2d) * m_cellSize + m_min;
    return make_float3(coord2d);
}

int3 CartesianCoordinates2D::getCellId3d(int cellId) const
{
    return int3{cellId%m_numGridCells.x, cellId/m_numGridCells.x,0};
}

int CartesianCoordinates2D::getCellId(const float3& coord) const
{
    int3 cellId3d = getCellId3d(coord);
    return cellId3d.y*m_numGridCells.x+ cellId3d.x;
}

int CartesianCoordinates2D::getCellId(const int3& cellId3d) const
{
    return cellId3d.y*m_numGridCells.x+ cellId3d.x;
}

int3 CartesianCoordinates2D::getCellId3d(const float3& coord) const
{
    float2 coord2d =  (make_float2(coord) - m_min) / m_cellSize;
    return make_int3(rintf(coord2d.x),rintf(coord2d.y),0);
}

int CartesianCoordinates2D::getRightNeighbor(int cellId) const
{
    return cellId+1;
}

int CartesianCoordinates2D::getLeftNeighbor(int cellId) const
{
    return cellId-1;
}

int CartesianCoordinates2D::getForwardNeighbor(int cellId) const
{
    return cellId+m_numGridCells.x;
}

int CartesianCoordinates2D::getBackwardNeighbor(int cellId) const
{
    return cellId-m_numGridCells.x;
}

int CartesianCoordinates2D::getUpNeighbor(int cellId) const
{
    return -1;
}

int CartesianCoordinates2D::getDownNeighbor(int cellId) const
{
    return -1;
}

int CartesianCoordinates2D::getNumGridCells() const
{
    return m_totalNumGridCells;
}

int3 CartesianCoordinates2D::getNumGridCells3d() const
{
    return make_int3(m_numGridCells,1);
}

int3 CartesianCoordinates2D::hasBoundary() const
{
    return make_int3(1,1,0);
}

float3 CartesianCoordinates2D::getCellSize() const
{
    return make_float3(m_cellSize);
}

float3 CartesianCoordinates2D::getMinCoord() const
{
    return float3{m_min.x, m_min.y, 0};
}

float3 CartesianCoordinates2D::getMaxCoord() const
{
    return float3{m_max.x,m_max.y,0};
}

int CartesianCoordinates2D::getDimension() const
{
    return 2;
}

int CartesianCoordinates2D::getCartesianDimension() const
{
    return 2;
}

float3 CartesianCoordinates2D::getAABBMin() const
{
    return float3{m_min.x, m_min.y, 0};
}

float3 CartesianCoordinates2D::getAABBMax() const
{
    return float3{m_max.x,m_max.y,0};
}

std::string CartesianCoordinates2D::getShaderDefine() const
{
    return "CARTESIAN_COORDINATES_2D";
}

void CartesianCoordinates2D::setShaderUniforms(mpu::gph::ShaderProgram& shader) const
{
    shader.uniform2f("csInternalData.m_min", glm::vec2(m_min.x,m_min.y));
    shader.uniform2f("csInternalData.m_max", glm::vec2(m_max.x,m_max.y));
    shader.uniform2f("csInternalData.m_size", glm::vec2(m_size.x,m_size.y));
    shader.uniform2f("csInternalData.m_cellSize", glm::vec2(m_cellSize.x,m_cellSize.y));
    shader.uniform2i("csInternalData.m_numGridCells", glm::ivec2(m_numGridCells.x,m_numGridCells.y));
    shader.uniform1i("csInternalData.m_totalNumGridCells", m_totalNumGridCells);
}

CSType CartesianCoordinates2D::getType()
{
    return CSType::cartesian2d;
}
