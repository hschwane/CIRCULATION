#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * CartesianCoordinates2D.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the CartesianCoordinates2D class
 *
 * Copyright (c) 2019 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "CartesianCoordinates2D.h"
//--------------------

// namespace
//--------------------

//--------------------

// function definitions of the CartesianCoordinates2D class
//-------------------------------------------------------------------
CartesianCoordinates2D::CartesianCoordinates2D(float3 min, float3 max, int3 numGridCells)
    : m_min(make_float2(min)), m_max(make_float2(max)), m_numGridCells(make_int2(numGridCells)),
    m_totalNumGridCells(numGridCells.x*numGridCells.y),
    m_size(m_max-m_min),
    m_cellSize( m_size / make_float2(m_numGridCells))
{
}

float3 CartesianCoordinates2D::getCartesian(const float3& coord) const
{
    return float3{coord.x,coord.y,0};
}

float3 CartesianCoordinates2D::getCoord(const float3& cartesian) const
{
    return float3{cartesian.x,cartesian.y,0};
}

float3 CartesianCoordinates2D::getCellCoordinate(int cellId) const
{
    return getCellCoordinate3d({cellId%m_numGridCells.x, cellId/m_numGridCells.x,0});
}

float3 CartesianCoordinates2D::getCellCoordinate3d(const int3& cellId3d) const
{
    int2 cellId2d = make_int2(cellId3d);
    float2 coord2d = make_float2(cellId2d) * m_cellSize + m_min;
    return make_float3(coord2d);
}

int CartesianCoordinates2D::getCellId(const float3& coord) const
{
    int3 cellId3d = getCellId3d(coord);
    return cellId3d.y*m_numGridCells.x+ cellId3d.x;
}

int3 CartesianCoordinates2D::getCellId3d(const float3& coord) const
{
    float2 coord2d =  (make_float2(coord) - m_min) / m_cellSize;
    return make_int3(rintf(coord2d.x),rintf(coord2d.y),0);
}

int CartesianCoordinates2D::getRightNeighbor(int cellId) const
{
    return cellId+1;
}

int CartesianCoordinates2D::getLeftNeighbor(int cellId) const
{
    return cellId-1;
}

int CartesianCoordinates2D::getForwardNeighbor(int cellId) const
{
    return cellId+m_numGridCells.x;
}

int CartesianCoordinates2D::getBackwardNeighbor(int cellId) const
{
    return cellId-m_numGridCells.x;
}

int CartesianCoordinates2D::getUpNeighbor(int cellId) const
{
    return -1;
}

int CartesianCoordinates2D::getDownNeighbor(int cellId) const
{
    return -1;
}

int CartesianCoordinates2D::numGridCells() const
{
    return m_totalNumGridCells;
}

int3 CartesianCoordinates2D::numGridCells3d() const
{
    return make_int3(m_numGridCells);
}

float3 CartesianCoordinates2D::getCellSize() const
{
    return make_float3(m_cellSize);
}

float3 CartesianCoordinates2D::getMinCoord() const
{
    return float3{m_min.x, m_min.y, 0};
}

float3 CartesianCoordinates2D::getMaxCoord() const
{
    return float3{m_max.x,m_max.y,0};
}

int CartesianCoordinates2D::getDimension() const
{
    return 2;
}

int CartesianCoordinates2D::getCartesianDimension() const
{
    return 2;
}

float3 CartesianCoordinates2D::getAABBMin() const
{
    return float3{m_min.x, m_min.y, 0};
}

float3 CartesianCoordinates2D::getAABBMax() const
{
    return float3{m_max.x,m_max.y,0};
}
