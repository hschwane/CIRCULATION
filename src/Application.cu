#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * Application.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the Application class
 *
 * Copyright (c) 2019 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "Application.h"
#include <random>
//--------------------

// function definitions of the Application class
//-------------------------------------------------------------------
Application::Application(int width, int height)
    : m_window(width,height,"CIRCULATION"),
    m_camera(mpu::gph::Camera::trackball, glm::vec3(0,0,2), glm::vec3(0,0,0)),
    m_renderer(width,height)
{
    // add shader include pathes
    mpu::gph::addShaderIncludePath(MPU_LIB_SHADER_PATH"include");
    mpu::gph::addShaderIncludePath(PROJECT_SHADER_PATH"include");

    // setup GUI
    ImGui::create(m_window);

    // some gl settings
    mpu::gph::enableVsync(m_vsync);

    // add resize callback
    m_window.addFBSizeCallback([this](int w, int h)
                             {
                                 glViewport(0,0,w,h);
                                 this->m_width = w;
                                 this->m_width = h;
                                 this->m_renderer.setSize(w,h);
                                 this->m_aspect = float(m_width) / float(m_height);
                             });

    // add input functions
    m_camera.addInputs();
    addInputs();
    setKeybindings();
}

bool Application::run()
{
    mpu::gph::Input::update();
    if( !m_window.frameBegin())
        return false;

    // -------------------------
    // handle user interface
    // draw main menu
    mainMenuBar();

    // draw windows if needed
    if(m_showImGuiDemoWindow) ImGui::ShowDemoWindow(&m_showImGuiDemoWindow);
    if(m_showCameraDebugWindow) m_camera.showDebugWindow(&m_showCameraDebugWindow);
    if(m_showPerfWindow) showPerfWindow(&m_showPerfWindow);
    if(m_showAboutWindow) showAboutWindow(&m_showAboutWindow);
    if(m_showKeybindingsWindow) showKeybindingsWindow(&m_showKeybindingsWindow);
    if(m_showRendererWindow) m_renderer.showGui(&m_showRendererWindow);

    // open new simulation modal on startup
    static struct Once{Once(){ImGui::OpenPopup("New Simulation");}}once;
    newSimulationModal();

    // -------------------------
    // simulation

    // -------------------------
    // rendering
    m_camera.update();
    m_renderer.setViewMat(m_camera.viewMatrix());
    m_renderer.draw();

    m_window.frameEnd();
    return true;
}

void Application::addInputs()
{
    using namespace mpu::gph;
    // close app on escape
    Input::addButton("Close", "close application",
                          [](Window& wnd) { wnd.shouldClose(); });

    // fullscreen app on F11
    Input::addButton("ToggleFullscreen","switch between fullscreen and windowed mode",
                          [](Window& wnd) { wnd.toggleFullscreen(); });

    // ability to reset the camera
    Input::addButton("ResetCamera", "reset the camera based on loaded grid",
            [this](Window&) { this->resetCamera(); });

    // hde gui for nice screenshots
    Input::addButton("ToggleGUI","toggle visibility the user interface", [this](Window&){ImGui::toggleVisibility();});
}

void Application::setKeybindings()
{
    using namespace mpu::gph;

    // camera
    Input::mapKeyToInput("CameraMoveSideways",GLFW_KEY_D,Input::ButtonBehavior::whenDown,Input::AxisBehavior::positive);
    Input::mapKeyToInput("CameraMoveSideways",GLFW_KEY_A,Input::ButtonBehavior::whenDown,Input::AxisBehavior::negative);
    Input::mapKeyToInput("CameraMoveForwardBackward",GLFW_KEY_W,Input::ButtonBehavior::whenDown,Input::AxisBehavior::positive);
    Input::mapKeyToInput("CameraMoveForwardBackward",GLFW_KEY_S,Input::ButtonBehavior::whenDown,Input::AxisBehavior::negative);
    Input::mapKeyToInput("CameraMoveUpDown",GLFW_KEY_Q,Input::ButtonBehavior::whenDown,Input::AxisBehavior::negative);
    Input::mapKeyToInput("CameraMoveUpDown",GLFW_KEY_E,Input::ButtonBehavior::whenDown,Input::AxisBehavior::positive);

    Input::mapCourserToInput("CameraPanHorizontal", Input::AxisOrientation::horizontal,Input::AxisBehavior::negative,0, "EnablePan");
    Input::mapCourserToInput("CameraPanVertical", Input::AxisOrientation::vertical,Input::AxisBehavior::positive,0, "EnablePan");
    Input::mapScrollToInput("CameraZoom");

    Input::mapMouseButtonToInput("EnablePan", GLFW_MOUSE_BUTTON_MIDDLE);
    Input::mapKeyToInput("EnablePan", GLFW_KEY_LEFT_ALT);

    Input::mapCourserToInput("CameraRotateHorizontal", Input::AxisOrientation::horizontal,Input::AxisBehavior::negative,0, "EnableRotation");
    Input::mapCourserToInput("CameraRotateVertical", Input::AxisOrientation::vertical,Input::AxisBehavior::negative,0, "EnableRotation");

    Input::mapMouseButtonToInput("EnableRotation", GLFW_MOUSE_BUTTON_LEFT);
    Input::mapKeyToInput("EnableRotation", GLFW_KEY_LEFT_CONTROL);

    Input::mapKeyToInput("CameraMovementSpeed",GLFW_KEY_RIGHT_BRACKET,Input::ButtonBehavior::whenDown,Input::AxisBehavior::positive);
    Input::mapKeyToInput("CameraMovementSpeed",GLFW_KEY_SLASH,Input::ButtonBehavior::whenDown,Input::AxisBehavior::negative);
    Input::mapKeyToInput("CameraToggleMode",GLFW_KEY_R);
    Input::mapKeyToInput("CameraSlowMode",GLFW_KEY_LEFT_SHIFT,Input::ButtonBehavior::whenDown);
    Input::mapKeyToInput("CameraFastMode",GLFW_KEY_SPACE,Input::ButtonBehavior::whenDown);

    // generic
    Input::mapKeyToInput("Close",GLFW_KEY_ESCAPE);
    Input::mapKeyToInput("ToggleFullscreen",GLFW_KEY_F11);
    Input::mapKeyToInput("ResetCamera",GLFW_KEY_X);
    Input::mapKeyToInput("ToggleGUI", GLFW_KEY_TAB);
}

void Application::resetCamera()
{
    glm::vec3 aabbMin{m_currentCS->getAABBMin().x,m_currentCS->getAABBMin().y, m_currentCS->getAABBMin().z};
    glm::vec3 aabbMax{m_currentCS->getAABBMax().x,m_currentCS->getAABBMax().y, m_currentCS->getAABBMax().z};

    glm::vec3 size = aabbMax - aabbMin;
    float diagonal = glm::length(size);
    glm::vec3 center = aabbMin + size/2;

    m_camera.setPosition(glm::vec3(diagonal));
    m_camera.setTarget(center);
}

void Application::mainMenuBar()
{
    bool newSimPressed=false; // was Simulation -> New selected?

    if(ImGui::BeginMainMenuBar())
    {
        // simulation menu to manage the simulation
        if(ImGui::BeginMenu("Simulation"))
        {
            if(ImGui::MenuItem("New"))
                newSimPressed=true; // needed for some imGui id stack thing
            ImGui::EndMenu();
        }

        if(ImGui::BeginMenu("Visualization"))
        {
            ImGui::MenuItem("Show Visualization window", nullptr, &m_showRendererWindow);
            ImGui::Separator();

            if(ImGui::MenuItem("Reset Camera","X"))
                resetCamera();

            if(ImGui::MenuItem("Toggle Camera Mode","R"))
                m_camera.toggleMode();

            ImGui::EndMenu();
        }

        // window menu to select shown windows
        if(ImGui::BeginMenu("Windows"))
        {
            ImGui::MenuItem("performance", nullptr, &m_showPerfWindow);
            ImGui::MenuItem("visualization", nullptr, &m_showRendererWindow);
            ImGui::MenuItem("camera debug window", nullptr, &m_showCameraDebugWindow);
            ImGui::Separator();
            ImGui::MenuItem("ImGui demo window", nullptr, &m_showImGuiDemoWindow);
            ImGui::EndMenu();
        }

        // window menu to select shown windows
        if(ImGui::BeginMenu("Help"))
        {
            ImGui::MenuItem("Keybindings", nullptr, &m_showKeybindingsWindow);
            ImGui::MenuItem("About", nullptr, &m_showAboutWindow);
            ImGui::EndMenu();
        }

        ImGui::EndMenuBar();
    }

    // open modal
    if(newSimPressed)
        ImGui::OpenPopup("New Simulation");
}

void Application::showPerfWindow(bool* show)
{
    ImGui::SetNextWindowSize({0,0},ImGuiCond_FirstUseEver);
    if(ImGui::Begin("performance",show))
    {
        ImGui::Text("Frametime: %f", mpu::gph::Input::deltaTime());
        ImGui::Text("FPS: %f", 1.0f / mpu::gph::Input::deltaTime());

        if(ImGui::Checkbox("V-Sync",&m_vsync))
            mpu::gph::enableVsync(m_vsync);
    }
}

void Application::showAboutWindow(bool* show)
{
    ImGui::SetNextWindowPos(ImVec2(ImGui::GetIO().DisplaySize.x * 0.5f, ImGui::GetIO().DisplaySize.y * 0.5f),
                            ImGuiCond_Appearing, ImVec2(0.5f,0.5f));
    ImGui::SetNextWindowSize({500,0});
    if(ImGui::Begin("About",show, ImGuiWindowFlags_NoScrollbar | ImGuiWindowFlags_NoResize | ImGuiWindowFlags_NoCollapse))
    {
        ImGui::Text("CIRCULATION");
        ImGui::Text("Cuda InteRactive Climate simULATION");
        ImGui::Text("Developed by Hendrik Schwanekamp\nhendrik.schwanekamp@gmx.net");
        ImGui::Text("on Gituhb:\n https://github.com/hschwane/CIRCULATION");

        ImGui::Spacing();
        ImGui::Spacing();
        ImGui::Spacing();
        ImGui::Separator();

        ImGui::TextWrapped("Included third party software:\n\n "
                    "GCE-Math: A C++ generalized constant expression-based math library Copyright 2016-2019 Keith O'Hara This product includes software developed by Keith O'Hara (http://www.kthohr.com)\n"
                    "\n"
                    "This software contains source code provided by NVIDIA Corporation.\n\n"
                    "CUB by nvlabs (https://nvlabs.github.io/cub/)\n"
                    "\n"
                    "stb_image (https://github.com/nothings/stb) This software contains source code provided by Sean T. Barrett.\n"
                    "\n"
                    "Dear ImGui (https://github.com/ocornut/imgui) This software contains source code provided by Omar Cornut.\n"
                    "\n"
                    "tiny file dialogs (ysengrin.com) This software contains source code provided by Guillaume Vareille.\n"
                    "\n"
                    "Test textures by Thomas Schmall (https://www.oxpal.com/uv-checker-texture.html)\n"
                    "\n"
                    "GLShader by Johannes Braun (https://github.com/johannes-braun/GLshader)  \n");

        ImGui::Separator();
        if(ImGui::Button("Close"))
            *show = false;
    }
}

void Application::showKeybindingsWindow(bool* show)
{
    ImGui::SetNextWindowSize({0,0},ImGuiCond_FirstUseEver);
    ImGui::SetNextWindowPos(ImVec2(ImGui::GetIO().DisplaySize.x * 0.5f, ImGui::GetIO().DisplaySize.y * 0.5f),
                            ImGuiCond_Appearing, ImVec2(0.5f,0.5f));
    if(ImGui::Begin("Keybindings",show))
    {
        ImGui::Text("Keybindings on german keyboard:");

        if(ImGui::CollapsingHeader("General"))
        {
            ImGui::Columns(2);
            ImGui::Text("ESC"); ImGui::NextColumn(); ImGui::Text("Close Application"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("F11"); ImGui::NextColumn(); ImGui::Text("Toggle Fullscreen"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("TAB"); ImGui::NextColumn(); ImGui::Text("Toggle User Interface"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Columns(1);
        }

        if(ImGui::CollapsingHeader("Camera"))
        {
            ImGui::Columns(2);
            ImGui::Text("Left MB or CTRL + mouse"); ImGui::NextColumn(); ImGui::Text("Rotate"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("WASD"); ImGui::NextColumn(); ImGui::Text("Move"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("Q/E"); ImGui::NextColumn(); ImGui::Text("Move up / down"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("Middle MB or ALT + mouse"); ImGui::NextColumn(); ImGui::Text("Pan"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("Middle Wheel"); ImGui::NextColumn(); ImGui::Text("Zoom"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("hold SHIFT"); ImGui::NextColumn(); ImGui::Text("Slower movement"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("hold SPACE"); ImGui::NextColumn(); ImGui::Text("Faster movement"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("+/-"); ImGui::NextColumn(); ImGui::Text("increase / decrease movement speed"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("R"); ImGui::NextColumn(); ImGui::Text("switch between \"trackball\" and \"first person\" movement"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("X"); ImGui::NextColumn(); ImGui::Text("reset camera position and orientation"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Columns(1);
        }

        ImGui::Separator();
        if(ImGui::Button("Close"))
            *show = false;
    }
}

void Application::newSimulationModal()
{
    if(ImGui::BeginPopupModal("New Simulation",nullptr,ImGuiWindowFlags_AlwaysAutoResize))
    {
        static int selctedModel = 0;
        static int selctedCoordinates = 0;
        static int3 numGridCells{128,128,32};
        static float3 minCoords{-1,-1,-1};
        static float3 maxCoords{1,1,1};

        // select simulation model and coordinate system
        ImGui::Combo("Model",&selctedModel,"Render Demo\0\0");
        ImGui::Combo("Coordinate System",&selctedCoordinates,"2D Cartesian Coordinate\0\0");

        // figure out dimension
        auto cs = coordinateSystemFactory(static_cast<CSType>(selctedCoordinates),{0,0,0},{0,0,0},{0,0,0});

        // depending on the dimension number of selected system
        if(cs->getDimension() == 2)
        {
            ImGui::DragInt2("Number of Grid Cells", &numGridCells.x);
            ImGui::DragFloat2("Min coordinates", &minCoords.x);
            ImGui::DragFloat2("Max coordinates", &maxCoords.x);

            float3 size = maxCoords - minCoords;
            float3 cellSize = size / make_float3(numGridCells);
            ImGui::PushItemFlag(ImGuiItemFlags_Disabled, true);
            ImGui::PushStyleVar(ImGuiStyleVar_Alpha, ImGui::GetStyle().Alpha * 0.5f);
            ImGui::DragFloat2("Size", &size.x);
            ImGui::DragFloat2("Cell Size", &cellSize.x);
            int numOfCells = numGridCells.x * numGridCells.y;
            ImGui::DragInt("Total number of cells", &numOfCells);
            ImGui::PopItemFlag();
            ImGui::PopStyleVar();
        }
        else
        {
            ImGui::DragInt3("Number of Grid Cells", &numGridCells.x);
            ImGui::DragFloat3("Min coordinates", &minCoords.x);
            ImGui::DragFloat3("Max coordinates", &maxCoords.x);

            float3 size = maxCoords - minCoords;
            float3 cellSize = size / make_float3(numGridCells);
            ImGui::PushItemFlag(ImGuiItemFlags_Disabled, true);
            ImGui::PushStyleVar(ImGuiStyleVar_Alpha, ImGui::GetStyle().Alpha * 0.5f);
            ImGui::DragFloat3("Size", &size.x);
            ImGui::DragFloat3("Cell Size", &cellSize.x);
            ImGui::PopItemFlag();
            ImGui::PopStyleVar();
        }

        if(ImGui::Button("Cancel"))
            ImGui::CloseCurrentPopup();
        ImGui::SameLine();

        if(ImGui::Button("Create"))
        {
            ImGui::CloseCurrentPopup();

            if(cs->getDimension() == 2)
            {
                minCoords.z=0;
                maxCoords.z=0;
                numGridCells.z=0;
            }

            createNewSim(static_cast<SimModel>(selctedModel), static_cast<CSType>(selctedCoordinates), minCoords, maxCoords, numGridCells);
        }
        ImGui::SetItemDefaultFocus();

        ImGui::EndPopup();
    }
}

void Application::createNewSim(SimModel model, CSType coordinateSystem, const float3& min, const float3& max, const int3& cells)
{
    logINFO("Application") << "Creating new simulation with sim model " << int(model) << " coordinate system "
                           << int(coordinateSystem) << " coordinate range [" << min << "|" << max << "] and grid cell count " << cells;

    m_currentCS = coordinateSystemFactory(coordinateSystem, min, max, cells);
    m_renderer.setCS(m_currentCS);

    switch(model)
    {
        case SimModel::renderDemo:
        {
            RenderDemoGrid(m_currentCS->getNumGridCells());
            generateDemoData(m_demoGrid);
            m_demoGrid.addRenderBufferToVao(m_renderer.getVAO(), 0);
            m_demoGrid.bindRenderBuffer(0, GL_SHADER_STORAGE_BUFFER);
            break;
        }
    }

    resetCamera();
}

void Application::generateDemoData(RenderDemoGrid& grid)
{
    std::default_random_engine rng(mpu::getRanndomSeed());
    std::normal_distribution<float> dist(10,4);

    for(int i : mpu::Range<int>(grid.size()))
    {
        float density = fmax(0,dist(rng));
        float velX = fmax(0,dist(rng));
        float velY = fmax(0,dist(rng));

        grid.write<AT::density>(i,density);
        grid.write<AT::velocityX>(i,velX);
        grid.write<AT::velocityY>(i,velY);
    }

    grid.swapAndRender();
}
