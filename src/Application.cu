#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * Application.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the Application class
 *
 * Copyright (c) 2019 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "Application.h"
#include <random>
//--------------------

// function definitions of the Application class
//-------------------------------------------------------------------
Application::Application(int width, int height)
    : m_window(width,height,"CIRCULATION"),
    m_camera(mpu::gph::Camera::trackball, glm::vec3(0,0,2), glm::vec3(0,0,0),glm::vec3(0,0,1)),
    m_renderer(width,height)
{
    // setup GUI
    ImGui::create(m_window);

    // some gl settings
    mpu::gph::enableVsync(m_vsync);

    // add resize callback
    m_window.addFBSizeCallback([this](int w, int h)
                             {
                                 glViewport(0,0,w,h);
                                 this->m_width = w;
                                 this->m_width = h;
                                 this->m_renderer.setSize(w,h);
                                 this->m_aspect = float(m_width) / float(m_height);
                             });

    // add input functions
    m_camera.setZoomSpeed(0.2);
    m_camera.addInputs();
    addInputs();
    setKeybindings();
}

bool Application::run()
{
    mpu::gph::Input::update();
    if( !m_window.frameBegin())
        return false;

    // -------------------------
    // handle user interface
    // draw main menu
    mainMenuBar();

    // draw windows if needed
    if(m_showImGuiDemoWindow) ImGui::ShowDemoWindow(&m_showImGuiDemoWindow);
    if(m_showCameraDebugWindow) m_camera.showDebugWindow(&m_showCameraDebugWindow);
    if(m_showPerfWindow) showPerfWindow(&m_showPerfWindow);
    if(m_showAboutWindow) showAboutWindow(&m_showAboutWindow);
    if(m_showKeybindingsWindow) showKeybindingsWindow(&m_showKeybindingsWindow);
    if(m_showRendererWindow) m_renderer.showGui(&m_showRendererWindow);
    if(m_showSimulationWindow && m_simulation != nullptr) m_simulation->showGui(&m_showSimulationWindow);

    // open new simulation modal on startup
    static struct Once{Once(){ImGui::OpenPopup("New Simulation");}}once;
    newSimulationModal();

    // -------------------------
    // simulation
    if(m_simulation)
        m_simulation->run();

    // -------------------------
    // rendering
    m_camera.update();
    m_renderer.setViewMat(m_camera.viewMatrix());
    if(m_grid)
    {
        m_grid->startRendering();
        m_renderer.draw();
        m_grid->renderDone();
    }

    m_window.frameEnd();
    return true;
}

void Application::addInputs()
{
    using namespace mpu::gph;
    // close app on escape
    Input::addButton("Close", "close application",
                          [](Window& wnd) { wnd.shouldClose(); });

    // fullscreen app on F11
    Input::addButton("ToggleFullscreen","switch between fullscreen and windowed mode",
                          [](Window& wnd) { wnd.toggleFullscreen(); });

    // ability to reset the camera
    Input::addButton("ResetCamera", "reset the camera based on loaded grid",
            [this](Window&) { this->resetCamera(); });

    // hde gui for nice screenshots
    Input::addButton("ToggleGUI","toggle visibility the user interface", [](Window&){ImGui::toggleVisibility();});

    // add buttons to pause and resume
    Input::addButton("Pause", "Pause the simulation",  [this](Window& wnd) { if(this->m_simulation)this->m_simulation->pause(); });
    Input::addButton("Resume", "Resume the simulation",  [this](Window& wnd) { if(this->m_simulation)this->m_simulation->resume(); });
}

void Application::setKeybindings()
{
    using namespace mpu::gph;

    // camera
    Input::mapKeyToInput("CameraMoveSideways",GLFW_KEY_D,Input::ButtonBehavior::whenDown,Input::AxisBehavior::positive);
    Input::mapKeyToInput("CameraMoveSideways",GLFW_KEY_A,Input::ButtonBehavior::whenDown,Input::AxisBehavior::negative);
    Input::mapKeyToInput("CameraMoveForwardBackward",GLFW_KEY_W,Input::ButtonBehavior::whenDown,Input::AxisBehavior::positive);
    Input::mapKeyToInput("CameraMoveForwardBackward",GLFW_KEY_S,Input::ButtonBehavior::whenDown,Input::AxisBehavior::negative);
    Input::mapKeyToInput("CameraMoveUpDown",GLFW_KEY_Q,Input::ButtonBehavior::whenDown,Input::AxisBehavior::negative);
    Input::mapKeyToInput("CameraMoveUpDown",GLFW_KEY_E,Input::ButtonBehavior::whenDown,Input::AxisBehavior::positive);

    Input::mapCourserToInput("CameraPanHorizontal", Input::AxisOrientation::horizontal,Input::AxisBehavior::negative,0, "EnablePan");
    Input::mapCourserToInput("CameraPanVertical", Input::AxisOrientation::vertical,Input::AxisBehavior::positive,0, "EnablePan");
    Input::mapScrollToInput("CameraZoom");

    Input::mapMouseButtonToInput("EnablePan", GLFW_MOUSE_BUTTON_MIDDLE);
    Input::mapKeyToInput("EnablePan", GLFW_KEY_LEFT_ALT);

    Input::mapCourserToInput("CameraRotateHorizontal", Input::AxisOrientation::horizontal,Input::AxisBehavior::negative,0, "EnableRotation");
    Input::mapCourserToInput("CameraRotateVertical", Input::AxisOrientation::vertical,Input::AxisBehavior::negative,0, "EnableRotation");

    Input::mapMouseButtonToInput("EnableRotation", GLFW_MOUSE_BUTTON_LEFT);
    Input::mapKeyToInput("EnableRotation", GLFW_KEY_LEFT_CONTROL);

    Input::mapKeyToInput("CameraMovementSpeed",GLFW_KEY_RIGHT_BRACKET,Input::ButtonBehavior::whenDown,Input::AxisBehavior::positive);
    Input::mapKeyToInput("CameraMovementSpeed",GLFW_KEY_SLASH,Input::ButtonBehavior::whenDown,Input::AxisBehavior::negative);
    Input::mapKeyToInput("CameraToggleMode",GLFW_KEY_R);
    Input::mapKeyToInput("CameraSlowMode",GLFW_KEY_LEFT_SHIFT,Input::ButtonBehavior::whenDown);
    Input::mapKeyToInput("CameraFastMode",GLFW_KEY_SPACE,Input::ButtonBehavior::whenDown);

    // generic
    Input::mapKeyToInput("Close",GLFW_KEY_ESCAPE);
    Input::mapKeyToInput("ToggleFullscreen",GLFW_KEY_F11);
    Input::mapKeyToInput("ResetCamera",GLFW_KEY_X);
    Input::mapKeyToInput("ToggleGUI", GLFW_KEY_TAB);

    // simulation
    Input::mapKeyToInput("Resume",GLFW_KEY_1);
    Input::mapKeyToInput("Pause",GLFW_KEY_2);
}

void Application::resetCamera()
{
    glm::vec3 aabbMin{m_cs->getAABBMin().x, m_cs->getAABBMin().y, m_cs->getAABBMin().z};
    glm::vec3 aabbMax{m_cs->getAABBMax().x, m_cs->getAABBMax().y, m_cs->getAABBMax().z};

    glm::vec3 size = aabbMax - aabbMin;
    float diagonal = glm::length(size);
    glm::vec3 center = aabbMin + size/2;

    m_camera.setPosition(glm::vec3(0,-0.75*diagonal,diagonal));
    m_camera.setTarget(center);
}

void Application::mainMenuBar()
{
    bool newSimPressed=false; // was Simulation -> New selected?

    if(ImGui::BeginMainMenuBar())
    {
        // simulation menu to manage the simulation
        if(ImGui::BeginMenu("Simulation"))
        {
            if(ImGui::MenuItem("New"))
                newSimPressed=true; // needed for some imGui id stack thing

            // disable menue in case simulation is not valid
            if(!m_simulation)
            {
                ImGui::PushItemFlag(ImGuiItemFlags_Disabled, true);
                ImGui::PushStyleVar(ImGuiStyleVar_Alpha, ImGui::GetStyle().Alpha * 0.5f);
            }

            ImGui::Separator();

            if(m_simulation && m_simulation->isPaused())
            {
                if(ImGui::MenuItem("Resume","1"))
                    m_simulation->resume();
            }
            else
            {
                if(ImGui::MenuItem("Pause","2"))
                    m_simulation->pause();
            }

            if(ImGui::MenuItem("Reset"))
            {
                m_grid = m_simulation->recreate(m_cs);
                m_grid->addRenderBufferToVao(m_renderer.getVAO(), 0);
                m_grid->bindRenderBuffer(0, GL_SHADER_STORAGE_BUFFER);
            }

            ImGui::Separator();

            ImGui::MenuItem("Show Simulation window", nullptr, &m_showSimulationWindow);

            if(!m_simulation)
            {
                ImGui::PopItemFlag();
                ImGui::PopStyleVar();
            }

            ImGui::EndMenu();
        }

        if(ImGui::BeginMenu("Visualization"))
        {
            if(ImGui::MenuItem("Reset Camera","X"))
                resetCamera();

            if(ImGui::MenuItem("Toggle Camera Mode","R"))
                m_camera.toggleMode();

            ImGui::Separator();
            ImGui::MenuItem("Show Visualization window", nullptr, &m_showRendererWindow);

            ImGui::EndMenu();
        }

        // window menu to select shown windows
        if(ImGui::BeginMenu("Windows"))
        {
            ImGui::MenuItem("performance", nullptr, &m_showPerfWindow);
            ImGui::MenuItem("visualization", nullptr, &m_showRendererWindow);
            ImGui::MenuItem("simulation", nullptr, &m_showSimulationWindow);
            ImGui::MenuItem("camera debug window", nullptr, &m_showCameraDebugWindow);
            ImGui::Separator();
            ImGui::MenuItem("ImGui demo window", nullptr, &m_showImGuiDemoWindow);
            ImGui::EndMenu();
        }

        // window menu to select shown windows
        if(ImGui::BeginMenu("Help"))
        {
            ImGui::MenuItem("Keybindings", nullptr, &m_showKeybindingsWindow);
            ImGui::MenuItem("About", nullptr, &m_showAboutWindow);
            ImGui::EndMenu();
        }

        ImGui::EndMainMenuBar();
    }

    // open modal
    if(newSimPressed)
        ImGui::OpenPopup("New Simulation");
}

void Application::showPerfWindow(bool* show)
{
    ImGui::SetNextWindowSize({0,0},ImGuiCond_FirstUseEver);
    if(ImGui::Begin("performance",show))
    {
        ImGui::Text("Frametime: %f", mpu::gph::Input::deltaTime());
        ImGui::Text("FPS: %f", 1.0f / mpu::gph::Input::deltaTime());

        if(ImGui::Checkbox("V-Sync",&m_vsync))
            mpu::gph::enableVsync(m_vsync);
    }
    ImGui::End();
}

void Application::showAboutWindow(bool* show)
{
    ImGui::SetNextWindowPos(ImVec2(ImGui::GetIO().DisplaySize.x * 0.5f, ImGui::GetIO().DisplaySize.y * 0.5f),
                            ImGuiCond_Appearing, ImVec2(0.5f,0.5f));
    ImGui::SetNextWindowSize({500,0});
    if(ImGui::Begin("About",show, ImGuiWindowFlags_NoScrollbar | ImGuiWindowFlags_NoResize | ImGuiWindowFlags_NoCollapse))
    {
        ImGui::Text("CIRCULATION");
        ImGui::Text("Cuda InteRactive Climate simULATION");
        ImGui::Text("Developed by Hendrik Schwanekamp\nhendrik.schwanekamp@gmx.net");
        ImGui::Text("on Gituhb:\n https://github.com/hschwane/CIRCULATION");

        ImGui::Spacing();
        ImGui::Spacing();
        ImGui::Spacing();
        ImGui::Separator();

        ImGui::TextWrapped("Included third party software:\n\n "
                    "GCE-Math: A C++ generalized constant expression-based math library Copyright 2016-2019 Keith O'Hara This product includes software developed by Keith O'Hara (http://www.kthohr.com)\n"
                    "\n"
                    "This software contains source code provided by NVIDIA Corporation.\n\n"
                    "CUB by nvlabs (https://nvlabs.github.io/cub/)\n"
                    "\n"
                    "stb_image (https://github.com/nothings/stb) This software contains source code provided by Sean T. Barrett.\n"
                    "\n"
                    "Dear ImGui (https://github.com/ocornut/imgui) This software contains source code provided by Omar Cornut.\n"
                    "\n"
                    "tiny file dialogs (ysengrin.com) This software contains source code provided by Guillaume Vareille.\n"
                    "\n"
                    "Test textures by Thomas Schmall (https://www.oxpal.com/uv-checker-texture.html)\n"
                    "\n"
                    "GLShader by Johannes Braun (https://github.com/johannes-braun/GLshader)  \n");

        ImGui::Separator();
        if(ImGui::Button("Close"))
            *show = false;
    }
    ImGui::End();
}

void Application::showKeybindingsWindow(bool* show)
{
    ImGui::SetNextWindowSize({0,0},ImGuiCond_FirstUseEver);
    ImGui::SetNextWindowPos(ImVec2(ImGui::GetIO().DisplaySize.x * 0.5f, ImGui::GetIO().DisplaySize.y * 0.5f),
                            ImGuiCond_Appearing, ImVec2(0.5f,0.5f));
    if(ImGui::Begin("Keybindings",show))
    {
        ImGui::Text("Keybindings on german keyboard:");

        if(ImGui::CollapsingHeader("General"))
        {
            ImGui::Columns(2);
            ImGui::Text("ESC"); ImGui::NextColumn(); ImGui::Text("Close Application"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("F11"); ImGui::NextColumn(); ImGui::Text("Toggle Fullscreen"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("TAB"); ImGui::NextColumn(); ImGui::Text("Toggle User Interface"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Columns(1);
        }

        if(ImGui::CollapsingHeader("Simulation"))
        {
            ImGui::Columns(2);
            ImGui::Text("1"); ImGui::NextColumn(); ImGui::Text("Resume Simulation"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("2"); ImGui::NextColumn(); ImGui::Text("Pause Simulation"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Columns(1);
        }

        if(ImGui::CollapsingHeader("Camera"))
        {
            ImGui::Columns(2);
            ImGui::Text("Left MB or CTRL + mouse"); ImGui::NextColumn(); ImGui::Text("Rotate"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("WASD"); ImGui::NextColumn(); ImGui::Text("Move"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("Q/E"); ImGui::NextColumn(); ImGui::Text("Move up / down"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("Middle MB or ALT + mouse"); ImGui::NextColumn(); ImGui::Text("Pan"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("Middle Wheel"); ImGui::NextColumn(); ImGui::Text("Zoom"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("hold SHIFT"); ImGui::NextColumn(); ImGui::Text("Slower movement"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("hold SPACE"); ImGui::NextColumn(); ImGui::Text("Faster movement"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("+/-"); ImGui::NextColumn(); ImGui::Text("increase / decrease movement speed"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("R"); ImGui::NextColumn(); ImGui::Text("switch between \"trackball\" and \"first person\" movement"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Text("X"); ImGui::NextColumn(); ImGui::Text("reset camera position and orientation"); ImGui::NextColumn(); ImGui::Separator();
            ImGui::Columns(1);
        }

        ImGui::Separator();
        if(ImGui::Button("Close"))
            *show = false;
    }
    ImGui::End();
}

void Application::newSimulationModal()
{
    if(ImGui::BeginPopupModal("New Simulation",nullptr,ImGuiWindowFlags_AlwaysAutoResize))
    {
        static int selctedCoordinates = 0;
        static int3 numGridCells{128,128,32};

        // variables for cartesian grids
        static float3 minCoords{-1,-1,-1};
        static float3 maxCoords{1,1,1};

        // variables for geographical grids
        static float minLat{-1.57f};
        static float maxLat{1.57f};
        static float radius{1.0f};

        // variables to select a simulation
        static auto testSim = std::make_unique<TestSimulation>();
        static auto rdSim = std::make_unique<RenderDemoSimulation>();
        static Simulation* selectedeModel = rdSim.get();
        static int selctedModelId = 0;

        // select simulation model
        if( ImGui::Combo("Model", &selctedModelId, "Render Demo \0 Test Simulation \0\0") )
        {
            switch(static_cast<SimModel>(selctedModelId))
            {
                case SimModel::renderDemo:
                    selectedeModel = rdSim.get();
                    break;
                case SimModel::testSimulation:
                    selectedeModel = testSim.get();
                    break;
            }
        }

        // handle model settings
        selectedeModel->drawCreationOptions();
        ImGui::Separator();

        // select coordinate system
        ImGui::Combo("Coordinate System",&selctedCoordinates,"2D Cartesian Coordinates \0 2D Geographical Coordinates \0\0");

        // options depending on coordinate system
        switch(static_cast<CSType>(selctedCoordinates))
        {
            case CSType::cartesian2d:
            {
                ImGui::PushID("Cartesian2dOptions");
                ImGui::DragInt2("Number of Grid Cells", &numGridCells.x);
                ImGui::DragFloat2("Min coordinates", &minCoords.x);
                ImGui::DragFloat2("Max coordinates", &maxCoords.x);

                float2 size = make_float2(maxCoords - minCoords);
                float2 cellSize = size / make_float2( (numGridCells.x<2) ? 1 : numGridCells.x-1, (numGridCells.y<2) ? 1 : numGridCells.y-1);
                ImGui::PushItemFlag(ImGuiItemFlags_Disabled, true);
                ImGui::PushStyleVar(ImGuiStyleVar_Alpha, ImGui::GetStyle().Alpha * 0.5f);
                ImGui::DragFloat2("Size", &size.x);
                ImGui::DragFloat2("Cell Size", &cellSize.x);
                int numOfCells = numGridCells.x * numGridCells.y;
                ImGui::DragInt("Total number of cells", &numOfCells);
                ImGui::PopItemFlag();
                ImGui::PopStyleVar();
                ImGui::PopID();
                break;
            }
            case CSType::geographical2d:
            {
                ImGui::PushID("Geographical2dOptions");
                ImGui::DragInt2("Number of Grid Cells", &numGridCells.x);
                ImGui::DragFloat("Min latitude", &minLat,0.001);
                ImGui::DragFloat("Max latitude", &maxLat,0.001);
                ImGui::DragFloat("Radius", &radius);

                float2 size = make_float2(2* M_PIf32, maxLat) - make_float2(0,minLat);
                float2 cellSize = size / make_float2( numGridCells.x, (numGridCells.y<2) ? 1 : numGridCells.y-1);
                ImGui::PushItemFlag(ImGuiItemFlags_Disabled, true);
                ImGui::PushStyleVar(ImGuiStyleVar_Alpha, ImGui::GetStyle().Alpha * 0.5f);
                ImGui::DragFloat2("Angular size", &size.x);
                ImGui::DragFloat2("Angular cell Size", &cellSize.x);
                int numOfCells = numGridCells.x * numGridCells.y;
                ImGui::DragInt("Total number of cells", &numOfCells);
                ImGui::PopItemFlag();
                ImGui::PopStyleVar();
                ImGui::PopID();
                break;
            }
        }

        // cancel button
        if(ImGui::Button("Cancel"))
            ImGui::CloseCurrentPopup();
        ImGui::SameLine();

        // create button
        if(ImGui::Button("Create"))
        {
            ImGui::CloseCurrentPopup();

            logINFO("Application") << "Creating new simulation with sim model " << int(selctedModelId) << " coordinate system "
                                   << int(selctedCoordinates) << "] and grid cell count " << numGridCells;

            // create coordinate system
            switch(static_cast<CSType>(selctedCoordinates))
            {
                case CSType::cartesian2d:
                {
                    minCoords.z = 0;
                    maxCoords.z = 0;
                    numGridCells.z = 0;
                    m_cs = std::make_shared<CartesianCoordinates2D>(minCoords, maxCoords, numGridCells);
                    break;
                }
                case CSType::geographical2d:
                {
                    m_cs = std::make_shared<GeographicalCoordinates2D>(minLat, maxLat, numGridCells, radius);
                }
            }
            m_renderer.setCS(m_cs);

            // create simulation and grid
            m_simulation = selectedeModel->clone();
            m_grid = m_simulation->recreate(m_cs);
            m_simulation->pause();

            // setup visualization
            switch(static_cast<SimModel>(selctedModelId))
            {
                case SimModel::renderDemo:
                {
                    m_grid->addRenderBufferToVao(m_renderer.getVAO(), 0);
                    m_grid->bindRenderBuffer(0, GL_SHADER_STORAGE_BUFFER);

                    std::vector<std::pair<std::string,int>> scalarFields;
                    scalarFields.emplace_back("density",0);
                    scalarFields.emplace_back("velocity_x",1);
                    scalarFields.emplace_back("velocity_y",2);
                    m_renderer.setScalarFields(scalarFields);

                    std::vector<std::pair<std::string,std::pair<int,int>>> vectorFields;
                    vectorFields.emplace_back("velocity",std::pair<int,int>(1,2));
                    m_renderer.setVecFields(vectorFields);
                    break;
                }
                case SimModel::testSimulation:
                {
                    m_grid->addRenderBufferToVao(m_renderer.getVAO(), 0);
                    m_grid->bindRenderBuffer(0, GL_SHADER_STORAGE_BUFFER);

                    std::vector<std::pair<std::string,int>> scalarFields;
                    scalarFields.emplace_back("density",0);
                    scalarFields.emplace_back("density_laplace",5);
                    scalarFields.emplace_back("velocity_divergence",6);
                    scalarFields.emplace_back("velocity_curl",7);
                    scalarFields.emplace_back("temperature",8);
                    m_renderer.setScalarFields(scalarFields);

                    std::vector<std::pair<std::string,std::pair<int,int>>> vectorFields;
                    vectorFields.emplace_back("velocity",std::pair<int,int>(1,2));
                    vectorFields.emplace_back("density_gradient",std::pair<int,int>(3,4));
                    vectorFields.emplace_back("temperature_gradient",std::pair<int,int>(9,10));
                    m_renderer.setVecFields(vectorFields);
                    break;
                }
            }


            resetCamera();
        }
        ImGui::SetItemDefaultFocus();

        ImGui::EndPopup();
    }
}