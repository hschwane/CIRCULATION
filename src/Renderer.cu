#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * Renderer.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the Renderer class
 *
 * Copyright (c) 2019 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "Renderer.h"
//--------------------

// function definitions of the Renderer class
//-------------------------------------------------------------------

Renderer::Renderer(int w, int h)
{
    // add shader include pathes
    mpu::gph::addShaderIncludePath(MPU_LIB_SHADER_PATH"include");
    mpu::gph::addShaderIncludePath(PROJECT_SHADER_PATH"include");

    // add shader files
    m_icosphereShader.setShaderModule({PROJECT_SHADER_PATH"icosahedron.vert"});
    m_icosphereShader.setShaderModule({PROJECT_SHADER_PATH"icosahedron.frag"});

    // initial values for matrices
    m_aspect = float(w)/float(h);
    m_view = glm::mat4(1.0f);
    m_model = glm::mat4(1.0f);
    // projection mat is build when compiling shaders

    // try compiling shaders
    compileShader();

    // initial settings
    glEnable(GL_DEPTH_TEST);
    glClearColor( m_backgroundColor.x, m_backgroundColor.y, m_backgroundColor.z, 1.0f);
    glPointSize(5.0f);
    glPolygonMode(GL_FRONT_AND_BACK,GL_LINE);
}

void Renderer::showGui(bool* show)
{
    ImGui::SetNextWindowSize({0,0},ImGuiCond_FirstUseEver);
    if(ImGui::Begin("Visualization",show))
    {
        if(ImGui::CollapsingHeader("General"))
        {
            if(ImGui::ColorEdit3("Background",glm::value_ptr(m_backgroundColor)))
                glClearColor( m_backgroundColor.x, m_backgroundColor.y, m_backgroundColor.z, 1.0f);
            if(ImGui::DragFloat("Scale",&m_scale,0.01f,0.0001f,1000.0f))
            {
                setModelMat(glm::scale(glm::mat4(1.0f),glm::vec3(m_scale)));
                setClip(0.001,m_unscaledFar*m_scale);
            }

            if(ImGui::Checkbox("Hide back-faces",&m_backfaceCulling))
                setBackfaceCulling(m_backfaceCulling);

            if(ImGui::Button("Rebuild Shader"))
                compileShader();
        }
    }
    ImGui::End();
}

void Renderer::compileShader()
{
    logINFO("Renderer") << "recompiling all visualization shaders";

    try
    {
        m_icosphereShader.rebuild();
        setModelMat(m_model);
        setViewMat(m_view);
        rebuildProjectionMat();
        updateMVP();
    }
    catch (const std::runtime_error& e)
    {
        logERROR("Renderer") << "Shader compilation failed! Fix shader and try again.";
        int r = tinyfd_messageBox("Error","Shader compilation failed! Check shader and press \"ok\". \nSee log for more information.",
                "okcancel", "error",1);
        if(r == 0)
            throw e;
        else
            compileShader();
    }
}

void Renderer::setSize(int w, int h)
{
    m_aspect = float(w) / float(h);
    rebuildProjectionMat();
}

void Renderer::setClip(float near, float far)
{
    m_near = near;
    m_far = far;
    rebuildProjectionMat();
}

mpu::gph::VertexArray& Renderer::getVAO()
{
    return m_vao;
}

void Renderer::draw()
{
    m_vao.bind();

    // visualize scalar field
    if(m_renderIcosphere)
    {
        m_icosphereShader.use();
        glDrawElements(GL_TRIANGLES, m_numIndices, GL_UNSIGNED_INT, 0);
    }
}

void Renderer::setNumIndices(int numGridpoints)
{
    m_numIndices = numGridpoints;
}

mpu::gph::VertexArray& Renderer::getVao()
{
    return m_vao;
}

void Renderer::setViewMat(const glm::mat4& view)
{
    m_view = view;
    m_icosphereShader.uniformMat4("viewMat", m_view);
    updateMVP();
}

void Renderer::rebuildProjectionMat()
{
    m_projection = glm::perspective(glm::radians(m_fovy),m_aspect,m_near,m_far);
    m_icosphereShader.uniformMat4("projectionMat", m_projection);
    updateMVP();
}

void Renderer::setModelMat(const glm::mat4& m)
{
    m_model = m;
    m_icosphereShader.uniformMat4("modelMat", m_model);
    updateMVP();
}

void Renderer::updateMVP()
{
    m_icosphereShader.uniformMat4("modelViewProjectionMat", m_projection * m_view * m_model);
}

void Renderer::setBackfaceCulling(bool enable)
{
    m_backfaceCulling = enable;
    if(enable)
    {
        glEnable(GL_CULL_FACE);
        glCullFace(GL_BACK);
    }
    else
    {
        glDisable(GL_CULL_FACE);
    }

}