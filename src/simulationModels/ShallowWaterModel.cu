#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * ShallowWaterModel.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the ShallowWaterModel class
 *
 * Copyright (c) 2020 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "ShallowWaterModel.h"

#include <mpUtils/mpUtils.h>
#include <mpUtils/mpGraphics.h>
#include <mpUtils/mpCuda.h>

#include "../GridReference.h"
#include "../coordinateSystems/CartesianCoordinates2D.h"
#include "../coordinateSystems/GeographicalCoordinates2D.h"
#include "../finiteDifferences.h"
#include "../boundaryConditions.h"
//--------------------

// function definitions of the ShallowWaterModel class
//-------------------------------------------------------------------

void ShallowWaterModel::showCreationOptions()
{

}

void ShallowWaterModel::showBoundaryOptions(const CoordinateSystem& cs)
{
}

void ShallowWaterModel::showSimulationOptions()
{
    ImGui::DragFloat("Geopotential diffusion",&m_geopotDiffusion,0.00001f,0.00001,1.0,"%.5f");
    ImGui::Checkbox("Use Leapfrog",&m_useLeapfrog);
    ImGui::DragFloat("Timestep",&m_timestep,0.000001,0.000001f,1.0,"%.6f");
    ImGui::Text("Simulated Time units: %f", m_totalSimulatedTime);
}

std::shared_ptr<GridBase> ShallowWaterModel::recreate(std::shared_ptr<CoordinateSystem> cs)
{
    m_cs = cs;
    m_grid = std::make_shared<ShallowWaterGrid>(m_cs->getNumGridCells());
    m_phiPlusKBuffer.resize(m_cs->getNumGridCells());

    // select coordinate system
    switch(m_cs->getType())
    {
        case CSType::cartesian2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<CartesianCoordinates2D&>( *(this->m_cs)) ); };
            break;
        case CSType::geographical2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<GeographicalCoordinates2D&>( *(this->m_cs)) ); };
            break;
    }

    reset();
    return m_grid;
}

void ShallowWaterModel::reset()
{

    m_grid->cacheOverwrite();

    float3 center = m_cs->getMinCoord() + (m_cs->getMaxCoord() - m_cs->getMinCoord())*0.5f;
    logINFO("bla") << m_cs->getMinCoord() << m_cs->getMaxCoord() << center;
    // create initial conditions using gaussian
    for(int i : mpu::Range<int>(m_grid->size()))
    {
        float velX = 0.0f;
        float velY = 0.0f;

        float3 c = m_cs->getCellCoordinate(i);

        float geopotential = fmax(1, 0.1 * glm::gauss<float>(c.x,center.x, 0.1f) * glm::gauss<float>(c.y,center.y, 0.1f));


        m_grid->initialize<AT::geopotential>(i, geopotential);
        m_grid->initialize<AT::velocityX>(i, velX);
        m_grid->initialize<AT::velocityY>(i, velY);
    }
    m_grid->pushCachToDevice();

    // swap buffers and ready for rendering
    m_grid->swapAndRender();

    // reset simulation state
    m_totalSimulatedTime = 0.0f;
    m_firstTimestep = true;
}

std::unique_ptr<Simulation> ShallowWaterModel::clone() const
{
    return std::make_unique<ShallowWaterModel>(*this);
}

void ShallowWaterModel::simulateOnce()
{
    m_simOnceFunc(); // calls correct template specialization
}

template <typename csT>
__global__ void shallowWaterSimulationA(ShallowWaterGrid::ReferenceType grid, csT coordinateSystem,
                                        mpu::VectorReference<float> phiPlusK, float timestep, bool useLeapfrog, float diffusion)
{
    csT cs = coordinateSystem;

    // updates geopotential for all non boundary cells
    // also calculates kinetic energy per unit mass
    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-cs.hasBoundary().y ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);
            float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

            // read values of quantities
            const float phi = grid.read<AT::geopotential>(cellId);
            const float velRightX = grid.read<AT::velocityX>(cellId);
            const float velForY   = grid.read<AT::velocityY>(cellId);
            const float velLeftX  = grid.read<AT::velocityX>(cs.getLeftNeighbor(cellId));
            const float velBackY  = grid.read<AT::velocityY>(cs.getBackwardNeighbor(cellId));

            // compute kinetic energy per unit mass
            const float velX = (velLeftX + velRightX) * 0.5f;
            const float velY = (velForY + velBackY) * 0.5f;
            const float kinEnergy = (velX * velX + velY * velY) * 0.5f;
            phiPlusK[cellId] = kinEnergy + phi;

            // compute geopotential advection time derivative dPhi/dt
            const float divv = divergence2d( velLeftX, velRightX, velBackY, velForY, cellPos, cs);
            float dphi_dt = -divv * phi;

            if(diffusion > 0)
            {
                const float phiLeft = grid.read<AT::geopotential>(cs.getLeftNeighbor(cellId));
                const float phiRight = grid.read<AT::geopotential>(cs.getRightNeighbor(cellId));
                const float phiFor = grid.read<AT::geopotential>(cs.getForwardNeighbor(cellId));
                const float phiBack = grid.read<AT::geopotential>(cs.getBackwardNeighbor(cellId));

                // compute geopotential diffusion
                const float lapphi = laplace2d(phiLeft,phiRight,phiBack,phiFor,phi,cellPos,cs);
                dphi_dt += diffusion * lapphi;
            }

            // compute values at t+1
            float nextPhi;
            if(useLeapfrog)
            {
                const float prevPhi = grid.readPrev<AT::geopotential>(cellId);
                nextPhi = prevPhi + dphi_dt * 2.0f*timestep;
            }
            else
                nextPhi = phi + dphi_dt * timestep;

            grid.write<AT::geopotential>(cellId,nextPhi);
        }
}

template <typename csT>
__global__ void shallowWaterSimulationB(ShallowWaterGrid::ReferenceType grid, csT coordinateSystem,
                                        mpu::VectorReference<const float> phiPlusK, float timestep, bool useLeapfrog)
{
    csT cs = coordinateSystem;

    // updates all non boundary velocities
    // TODO: handle velocities parallel to the boundary
    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-2*cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-2*cs.hasBoundary().y ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);
            float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

            // read values of quantities
            const float phiKRight = phiPlusK[cs.getRightNeighbor(cellId)];
            const float phiKForward = phiPlusK[cs.getForwardNeighbor(cellId)];
            const float phiK = phiPlusK[cellId];
            const float velX = grid.read<AT::velocityX>(cellId);
            const float velY = grid.read<AT::velocityY>(cellId);

            // compute dvX/dt and dvY/dt
            const float2 gradPhiK = gradient2d(phiK,phiKRight,phiK,phiKForward,cellPos,cs);
            const float dvX_dt = -gradPhiK.x;
            const float dvY_dt = -gradPhiK.y;

            // compute values at t+1
            float nextVelX;
            float nextVelY;
            if(useLeapfrog)
            {
                const float prevVelX = grid.readPrev<AT::velocityX>(cellId);
                const float prevVelY = grid.readPrev<AT::velocityY>(cellId);

                nextVelX = prevVelX + dvX_dt * 2.0f*timestep;
                nextVelY = prevVelY + dvY_dt * 2.0f*timestep;
            }
            else
            {
                nextVelX = velX + dvX_dt * timestep;
                nextVelY = velY + dvY_dt * timestep;
            }
            grid.write<AT::velocityX>(cellId,nextVelX);
            grid.write<AT::velocityY>(cellId,nextVelY);
        }
}


template <typename csT>
void ShallowWaterModel::simulateOnceImpl(csT& cs)
{
    dim3 blocksize{16,16,1};
    dim3 numBlocks{ static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().x ,blocksize.x)),
                    static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().y ,blocksize.y)), 1};

    shallowWaterSimulationA<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs,m_phiPlusKBuffer.getVectorReference(),
            m_timestep, !m_firstTimestep && m_useLeapfrog, m_geopotDiffusion);
    shallowWaterSimulationB<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs,m_phiPlusKBuffer.getVectorReference(),
            m_timestep, !m_firstTimestep && m_useLeapfrog);

    m_totalSimulatedTime += m_timestep;
    m_firstTimestep = false;
}

GridBase& ShallowWaterModel::getGrid()
{
    return *m_grid;
}

std::string ShallowWaterModel::getDisplayName()
{
    return "Shallow Water Model";
}