#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * ShallowWaterModel.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the ShallowWaterModel class
 *
 * Copyright (c) 2020 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "ShallowWaterModel.h"
#include "../GridReference.h"
#include "../coordinateSystems/CartesianCoordinates2D.h"
#include "../coordinateSystems/GeographicalCoordinates2D.h"
#include "../finiteDifferences.h"
#include "../boundaryConditions.h"
//--------------------

// function definitions of the ShallowWaterModel class
//-------------------------------------------------------------------

void ShallowWaterModel::showCreationOptions()
{

}

void ShallowWaterModel::showBoundaryOptions(const CoordinateSystem& cs)
{

}

void ShallowWaterModel::showSimulationOptions()
{

}

std::shared_ptr<GridBase> ShallowWaterModel::recreate(std::shared_ptr<CoordinateSystem> cs)
{
    m_cs = cs;
    m_grid = std::make_shared<ShallowWaterGrid>(m_cs->getNumGridCells());

    // select coordinate system
    switch(m_cs->getType())
    {
        case CSType::cartesian2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<CartesianCoordinates2D&>( *(this->m_cs)) ); };
            break;
        case CSType::geographical2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<GeographicalCoordinates2D&>( *(this->m_cs)) ); };
            break;
    }

    reset();
    return m_grid;
}

void ShallowWaterModel::reset()
{
    for(int i : mpu::Range<int>(m_grid->size()))
    {
        float velX = 0.0f;
        float velY = 0.0f;
        float geopotential = 10.0f;

        m_grid->initialize<AT::geopotential>(i,geopotential);
        m_grid->initialize<AT::velocityX>(i, velX);
        m_grid->initialize<AT::velocityY>(i, velY);
    }
}

std::unique_ptr<Simulation> ShallowWaterModel::clone() const
{
    return std::make_unique<ShallowWaterModel>(*this);
}

void ShallowWaterModel::simulateOnce()
{
    m_simOnceFunc(); // calls correct template specialization
}

template <typename csT>
__global__ void shallowWaterSimulation(ShallowWaterGrid::ReferenceType grid, csT coordinateSystem)
{
    csT cs = coordinateSystem;

    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-cs.hasBoundary().y ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);
            float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );


        }
}


template <typename csT>
void ShallowWaterModel::simulateOnceImpl(csT& cs)
{
    dim3 blocksize{16,16,1};
    dim3 numBlocks{ static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().x ,blocksize.x)),
                    static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().y ,blocksize.y)), 1};

    shallowWaterSimulation<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs);
}

GridBase& ShallowWaterModel::getGrid()
{
    return *m_grid;
}

std::string ShallowWaterModel::getDisplayName()
{
    return "Shallow Water Model";
}