#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * ShallowWaterModel.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the ShallowWaterModel class
 *
 * Copyright (c) 2020 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include <mpUtils/external/imgui/imgui_internal.h>
#include "ShallowWaterModel.h"
#include "../GridReference.h"
#include "../coordinateSystems/CartesianCoordinates2D.h"
#include "../coordinateSystems/GeographicalCoordinates2D.h"
#include "../finiteDifferences.h"
#include "../boundaryConditions.h"
//--------------------

// function definitions of the ShallowWaterModel class
//-------------------------------------------------------------------

void ShallowWaterModel::showCreationOptions()
{

}

void ShallowWaterModel::showBoundaryOptions(const CoordinateSystem& cs)
{

}

void ShallowWaterModel::showSimulationOptions()
{
    ImGui::DragFloat("Timestep",&m_timestep,0.0001,0.0001f,1.0,"%.4f");
    ImGui::Text("Simulated Time units: %f", m_totalSimulatedTime);
}

std::shared_ptr<GridBase> ShallowWaterModel::recreate(std::shared_ptr<CoordinateSystem> cs)
{
    m_cs = cs;
    m_grid = std::make_shared<ShallowWaterGrid>(m_cs->getNumGridCells());

    // select coordinate system
    switch(m_cs->getType())
    {
        case CSType::cartesian2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<CartesianCoordinates2D&>( *(this->m_cs)) ); };
            break;
        case CSType::geographical2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<GeographicalCoordinates2D&>( *(this->m_cs)) ); };
            break;
    }

    reset();
    return m_grid;
}

void ShallowWaterModel::reset()
{
    // create initial conditions
    for(int i : mpu::Range<int>(m_grid->size()))
    {
        float velX = 0.0f;
        float velY = 0.0f;
        float geopotential = 10.0f;

        m_grid->initialize<AT::geopotential>(i, geopotential);
        m_grid->initialize<AT::velocityX>(i, velX);
        m_grid->initialize<AT::velocityY>(i, velY);
    }

    // swap buffers and ready for rendering
    m_grid->swapAndRender();

    // reset simulation state
    m_totalSimulatedTime = 0.0f;
}

std::unique_ptr<Simulation> ShallowWaterModel::clone() const
{
    return std::make_unique<ShallowWaterModel>(*this);
}

void ShallowWaterModel::simulateOnce()
{
    m_simOnceFunc(); // calls correct template specialization
}

template <typename csT>
__global__ void shallowWaterSimulation(ShallowWaterGrid::ReferenceType grid, csT coordinateSystem, float timestep)
{
    csT cs = coordinateSystem;

    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-cs.hasBoundary().y ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);
            float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

            // read values of quantities
            const float Phi = grid.read<AT::geopotential>(cellId);
            const float velRightX = grid.read<AT::velocityX>(cellId);
            const float velForY   = grid.read<AT::velocityY>(cellId);

            // compute geopotential time derivative dPhi/dt
            const float velLeftX  = grid.read<AT::velocityX>(cs.getLeftNeighbor(cellId));
            const float velBackY  = grid.read<AT::velocityY>(cs.getBackwardNeighbor(cellId));

            const float divv = divergence2d( velLeftX, velRightX, velBackY, velForY, cellPos, cs);
            const float dPhi_dt = -divv * Phi;

            // compute wind velocity time derivative

            // compute values at t+1
            const float nextPhi = Phi + dPhi_dt * timestep;
            grid.write<AT::geopotential>(cellId,nextPhi);
        }
}


template <typename csT>
void ShallowWaterModel::simulateOnceImpl(csT& cs)
{
    dim3 blocksize{16,16,1};
    dim3 numBlocks{ static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().x ,blocksize.x)),
                    static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().y ,blocksize.y)), 1};

    shallowWaterSimulation<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs,m_timestep);
}

GridBase& ShallowWaterModel::getGrid()
{
    return *m_grid;
}

std::string ShallowWaterModel::getDisplayName()
{
    return "Shallow Water Model";
}