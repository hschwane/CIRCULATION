#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * ShallowWaterModel.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the ShallowWaterModel class
 *
 * Copyright (c) 2020 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "ShallowWaterModel.h"

#include <mpUtils/mpUtils.h>
#include <mpUtils/mpGraphics.h>
#include <mpUtils/mpCuda.h>

#include "../GridReference.h"
#include "../coordinateSystems/CartesianCoordinates2D.h"
#include "../coordinateSystems/GeographicalCoordinates2D.h"
#include "../finiteDifferences.h"
#include "../boundaryConditions.h"
//--------------------

// function definitions of the ShallowWaterModel class
//-------------------------------------------------------------------

void ShallowWaterModel::showCreationOptions()
{

}

void ShallowWaterModel::showBoundaryOptions(const CoordinateSystem& cs)
{

}

void ShallowWaterModel::showSimulationOptions()
{
    ImGui::DragFloat("Timestep",&m_timestep,0.000001,0.000001f,1.0,"%.6f");
    ImGui::Text("Simulated Time units: %f", m_totalSimulatedTime);
}

std::shared_ptr<GridBase> ShallowWaterModel::recreate(std::shared_ptr<CoordinateSystem> cs)
{
    m_cs = cs;
    m_grid = std::make_shared<ShallowWaterGrid>(m_cs->getNumGridCells());
    m_phiPlusKBuffer.resize(m_cs->getNumGridCells());

    // select coordinate system
    switch(m_cs->getType())
    {
        case CSType::cartesian2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<CartesianCoordinates2D&>( *(this->m_cs)) ); };
            break;
        case CSType::geographical2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<GeographicalCoordinates2D&>( *(this->m_cs)) ); };
            break;
    }

    reset();
    return m_grid;
}

void ShallowWaterModel::reset()
{
    // create initial conditions
    for(int i : mpu::Range<int>(m_grid->size()))
    {
        float velX = 0.0f;
        float velY = 0.0f;
        float geopotential = 1.0f;

        m_grid->initialize<AT::geopotential>(i, geopotential);
        m_grid->initialize<AT::velocityX>(i, velX);
        m_grid->initialize<AT::velocityY>(i, velY);
    }

    // initialize some cells to a higher potential
    float highPotential = 10.0f;
    int3 center = int3{ m_cs->getNumGridCells3d().x /2, m_cs->getNumGridCells3d().y /2, 0};
    int centerId = m_cs->getCellId(center);

    m_grid->initialize<AT::geopotential>( centerId, highPotential);
    m_grid->initialize<AT::geopotential>( m_cs->getLeftNeighbor(centerId), highPotential);
    m_grid->initialize<AT::geopotential>( m_cs->getRightNeighbor(centerId), highPotential);
    m_grid->initialize<AT::geopotential>( m_cs->getBackwardNeighbor(centerId), highPotential);
    m_grid->initialize<AT::geopotential>( m_cs->getForwardNeighbor(centerId), highPotential);

    // swap buffers and ready for rendering
    m_grid->swapAndRender();

    // reset simulation state
    m_totalSimulatedTime = 0.0f;
}

std::unique_ptr<Simulation> ShallowWaterModel::clone() const
{
    return std::make_unique<ShallowWaterModel>(*this);
}

void ShallowWaterModel::simulateOnce()
{
    m_simOnceFunc(); // calls correct template specialization
}

template <typename csT>
__global__ void shallowWaterSimulationA(ShallowWaterGrid::ReferenceType grid, csT coordinateSystem,
                                        mpu::VectorReference<float> phiPlusK, float timestep)
{
    csT cs = coordinateSystem;

    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-cs.hasBoundary().y ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);
            float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

            // read values of quantities
            const float phi = grid.read<AT::geopotential>(cellId);
            const float velRightX = grid.read<AT::velocityX>(cellId);
            const float velForY   = grid.read<AT::velocityY>(cellId);
            const float velLeftX  = grid.read<AT::velocityX>(cs.getLeftNeighbor(cellId));
            const float velBackY  = grid.read<AT::velocityY>(cs.getBackwardNeighbor(cellId));

            // compute kinetic energy per unit mass
            const float velX = (velLeftX + velRightX) * 0.5f;
            const float velY = (velForY + velBackY) * 0.5f;
            const float kinEnergy = (velX * velX + velY * velY) * 0.5f;
            phiPlusK[cellId] = kinEnergy + phi;

            // compute geopotential time derivative dPhi/dt
            const float divv = divergence2d( velLeftX, velRightX, velBackY, velForY, cellPos, cs);
            const float dphi_dt = -divv * phi;

            // compute values at t+1
            const float nextPhi = phi + dphi_dt * timestep;
            grid.write<AT::geopotential>(cellId,nextPhi);
        }
}

template <typename csT>
__global__ void shallowWaterSimulationB(ShallowWaterGrid::ReferenceType grid, csT coordinateSystem,
                                        mpu::VectorReference<const float> phiPlusK, float timestep)
{
    csT cs = coordinateSystem;

    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-2*cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-2*cs.hasBoundary().y ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);
            float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

            // read values of quantities
            const float phiKRight = phiPlusK[cs.getRightNeighbor(cellId)];
            const float phiKForward = phiPlusK[cs.getForwardNeighbor(cellId)];
            const float phiK = phiPlusK[cellId];
            const float velX = grid.read<AT::velocityX>(cellId);
            const float velY = grid.read<AT::velocityY>(cellId);

            // compute dvX/dt and dvY/dt
            const float2 gradPhiK = gradient2d(phiK,phiKRight,phiK,phiKForward,cellPos,cs);
            const float dvX_dt = -gradPhiK.x;
            const float dvY_dt = -gradPhiK.y;

            // compute values at t+1
            const float nextVelX = velX + dvX_dt * timestep;
            const float nextVelY = velY + dvY_dt * timestep;
            grid.write<AT::velocityX>(cellId,nextVelX);
            grid.write<AT::velocityY>(cellId,nextVelY);
        }
}


template <typename csT>
void ShallowWaterModel::simulateOnceImpl(csT& cs)
{
    dim3 blocksize{16,16,1};
    dim3 numBlocks{ static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().x ,blocksize.x)),
                    static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().y ,blocksize.y)), 1};

    shallowWaterSimulationA<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs,m_phiPlusKBuffer.getVectorReference(),m_timestep);
    shallowWaterSimulationB<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs,m_phiPlusKBuffer.getVectorReference(),m_timestep);

    m_totalSimulatedTime += m_timestep;
}

GridBase& ShallowWaterModel::getGrid()
{
    return *m_grid;
}

std::string ShallowWaterModel::getDisplayName()
{
    return "Shallow Water Model";
}