#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * ShallowWaterModel.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the ShallowWaterModel class
 *
 * Copyright (c) 2020 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "ShallowWaterModel.h"

#include <mpUtils/mpUtils.h>
#include <mpUtils/mpGraphics.h>
#include <mpUtils/mpCuda.h>

#include "../GridReference.h"
#include "../coordinateSystems/CartesianCoordinates2D.h"
#include "../coordinateSystems/GeographicalCoordinates2D.h"
#include "../finiteDifferences.h"
#include "../boundaryConditions.h"
//--------------------

// function definitions of the ShallowWaterModel class
//-------------------------------------------------------------------

void ShallowWaterModel::showCreationOptions()
{
    ImGui::DragFloat2("position of disturbance", &m_gaussianPosition.x, 0.001);
    ImGui::DragFloat("standard deviation", &m_stdDev,0.01f);
    ImGui::DragFloat("multiplier", &m_multiplier,0.01f);
}

void ShallowWaterModel::showBoundaryOptions(const CoordinateSystem& cs)
{
}

void ShallowWaterModel::showSimulationOptions()
{
    if(m_cs->getType() != CSType::geographical2d)
        ImGui::DragFloat("Coriolis parameter",&m_coriolisParameter,0.0001f,0.000001,5.0f,"%.7f");
    else
        ImGui::DragFloat("Angular Velocity",&m_angularVelocity,0.00001f,0.00001,5.0f,"%.5f");

    ImGui::DragFloat("Geopotential diffusion",&m_geopotDiffusion,0.00001f,0.00001,1.0,"%.5f");
    ImGui::Checkbox("Use Leapfrog",&m_useLeapfrog);
    ImGui::DragFloat("Timestep",&m_timestep,0.000001,0.000001f,1.0,"%.6f");
    ImGui::Text("Simulated Time units: %f", m_totalSimulatedTime);
}

std::shared_ptr<GridBase> ShallowWaterModel::recreate(std::shared_ptr<CoordinateSystem> cs)
{
    m_cs = cs;
    m_grid = std::make_shared<ShallowWaterGrid>(m_cs->getNumGridCells());
    m_phiPlusKBuffer.resize(m_cs->getNumGridCells());
    m_vortPlusCor.resize(m_cs->getNumGridCells());

    // select coordinate system
    switch(m_cs->getType())
    {
        case CSType::cartesian2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<CartesianCoordinates2D&>( *(this->m_cs)) ); };
            break;
        case CSType::geographical2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<GeographicalCoordinates2D&>( *(this->m_cs)) ); };
            break;
    }

    reset();
    return m_grid;
}

void ShallowWaterModel::reset()
{
    m_grid->cacheOverwrite();

    // create initial conditions using gaussian
    for(int i : mpu::Range<int>(m_grid->size()))
    {
        float velX = 0.0f;
        float velY = 0.0f;

        float3 c = m_cs->getCellCoordinate(i);

        float geopotential = fmax(1, m_multiplier * glm::gauss<float>(c.x,m_gaussianPosition.x, m_stdDev) * glm::gauss<float>(c.y,m_gaussianPosition.y, m_stdDev));

        m_grid->initialize<AT::geopotential>(i, geopotential);
        m_grid->initialize<AT::velocityX>(i, velX);
        m_grid->initialize<AT::velocityY>(i, velY);
    }
    m_grid->pushCachToDevice();

    // swap buffers and ready for rendering
    m_grid->swapAndRender();

    // reset simulation state
    m_totalSimulatedTime = 0.0f;
    m_firstTimestep = true;
}

std::unique_ptr<Simulation> ShallowWaterModel::clone() const
{
    return std::make_unique<ShallowWaterModel>(*this);
}

void ShallowWaterModel::simulateOnce()
{
    m_simOnceFunc(); // calls correct template specialization
}

template <typename csT>
__global__ void shallowWaterSimulationA(ShallowWaterGrid::ReferenceType grid, csT coordinateSystem,
                                        mpu::VectorReference<float> phiPlusK, mpu::VectorReference<float> vortPlusCor,
                                        float timestep, bool useLeapfrog, float diffusion, float corOrAngvel)
{
    csT cs = coordinateSystem;

    // updates geopotential for all non boundary cells
    // also calculates kinetic energy per unit mass
    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-cs.hasBoundary().y ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);
            float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

            // read values of quantities
            const float phi = grid.read<AT::geopotential>(cellId);
            const float velRightX = grid.read<AT::velocityX>(cellId);
            const float velForY   = grid.read<AT::velocityY>(cellId);
            const float velLeftX  = grid.read<AT::velocityX>(cs.getLeftNeighbor(cellId));
            const float velBackY  = grid.read<AT::velocityY>(cs.getBackwardNeighbor(cellId));
            const float velForX  = grid.read<AT::velocityX>(cs.getForwardNeighbor(cellId)); // used for vorticity
            const float velRightY  = grid.read<AT::velocityY>(cs.getRightNeighbor(cellId)); // used for vorticity

            // compute kinetic energy per unit mass
            const float velX = (velLeftX + velRightX) * 0.5f;
            const float velY = (velForY + velBackY) * 0.5f;
            const float kinEnergy = (velX * velX + velY * velY) * 0.5f;
            phiPlusK[cellId] = kinEnergy + phi;

            // calculate vorticity and coriolis parameter
            // if this looks strange consider where values are located on the C grid
            const float2 vortPos = cellPos + 0.5f * make_float2(cs.getCellSize()); // position where vorticity is computed
            const float vort = curl2d(velForY, velRightY, velRightX, velForX, vortPos, cs);
            float cor;
            if(cs.getType() == CSType::geographical2d)
                cor = 2*corOrAngvel*sin(vortPos.y);
            else if(cs.getType() == CSType::cartesian2d)
                cor = corOrAngvel;
            else
                cor = 0.0f;
            vortPlusCor[cellId] = vort + cor;

            // write potential vorticity
            grid.write<AT::potentialVort>(cellId, abs(vort+cor) / phi);

            // compute geopotential advection time derivative dPhi/dt
            const float divv = divergence2d( velLeftX, velRightX, velBackY, velForY, cellPos, cs);
            float dphi_dt = -divv * phi;

            if(diffusion > 0)
            {
                const float phiLeft = grid.read<AT::geopotential>(cs.getLeftNeighbor(cellId));
                const float phiRight = grid.read<AT::geopotential>(cs.getRightNeighbor(cellId));
                const float phiFor = grid.read<AT::geopotential>(cs.getForwardNeighbor(cellId));
                const float phiBack = grid.read<AT::geopotential>(cs.getBackwardNeighbor(cellId));

                // compute geopotential diffusion
                const float lapphi = laplace2d(phiLeft,phiRight,phiBack,phiFor,phi,cellPos,cs);
                dphi_dt += diffusion * lapphi;
            }

            // compute values at t+1
            float nextPhi;
            if(useLeapfrog)
            {
                const float prevPhi = grid.readPrev<AT::geopotential>(cellId);
                nextPhi = prevPhi + dphi_dt * 2.0f*timestep;
            }
            else
                nextPhi = phi + dphi_dt * timestep;

            grid.write<AT::geopotential>(cellId,nextPhi);
        }
}

template <typename csT>
__global__ void shallowWaterSimulationB(ShallowWaterGrid::ReferenceType grid, csT coordinateSystem,
                                        mpu::VectorReference<const float> phiPlusK, mpu::VectorReference<float> vortPlusCor,
                                        float timestep, bool useLeapfrog)
{
    csT cs = coordinateSystem;

    // updates all non boundary velocities
    // TODO: handle velocities parallel to the boundary
    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-2*cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-2*cs.hasBoundary().y ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);
            float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

            // read values of quantities
            const float phiKRight = phiPlusK[cs.getRightNeighbor(cellId)];
            const float phiKForward = phiPlusK[cs.getForwardNeighbor(cellId)];
            const float phiK = phiPlusK[cellId];
            const float vortCorLeft = vortPlusCor[cs.getLeftNeighbor(cellId)];
            const float vortCorBack = vortPlusCor[cs.getBackwardNeighbor(cellId)];
            const float vortCor = vortPlusCor[cellId];
            const float velX = grid.read<AT::velocityX>(cellId);
            const float velY = grid.read<AT::velocityY>(cellId);

            // compute dvX/dt and dvY/dt
            const float2 gradPhiK = gradient2d(phiK,phiKRight,phiK,phiKForward,cellPos,cs);
            const float dvX_dt = (vortCor+vortCorBack)*0.5f*velY -gradPhiK.x;
            const float dvY_dt = -(vortCor+vortCorLeft)*0.5f*velX -gradPhiK.y;

            // compute values at t+1
            float nextVelX;
            float nextVelY;
            if(useLeapfrog)
            {
                const float prevVelX = grid.readPrev<AT::velocityX>(cellId);
                const float prevVelY = grid.readPrev<AT::velocityY>(cellId);

                nextVelX = prevVelX + dvX_dt * 2.0f*timestep;
                nextVelY = prevVelY + dvY_dt * 2.0f*timestep;
            }
            else
            {
                nextVelX = velX + dvX_dt * timestep;
                nextVelY = velY + dvY_dt * timestep;
            }
            grid.write<AT::velocityX>(cellId,nextVelX);
            grid.write<AT::velocityY>(cellId,nextVelY);
        }
}


template <typename csT>
void ShallowWaterModel::simulateOnceImpl(csT& cs)
{
    dim3 blocksize{16,16,1};
    dim3 numBlocks{ static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().x ,blocksize.x)),
                    static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().y ,blocksize.y)), 1};

    shallowWaterSimulationA<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs,m_phiPlusKBuffer.getVectorReference(),
            m_vortPlusCor.getVectorReference(), m_timestep, !m_firstTimestep && m_useLeapfrog,
            m_geopotDiffusion, (m_cs->getType() == CSType::geographical2d) ? m_angularVelocity : m_coriolisParameter);
    shallowWaterSimulationB<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs,m_phiPlusKBuffer.getVectorReference(),
            m_vortPlusCor.getVectorReference(), m_timestep, !m_firstTimestep && m_useLeapfrog);

    m_totalSimulatedTime += m_timestep;
    m_firstTimestep = false;
}

GridBase& ShallowWaterModel::getGrid()
{
    return *m_grid;
}

std::string ShallowWaterModel::getDisplayName()
{
    return "Shallow Water Model";
}