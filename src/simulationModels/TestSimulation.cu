#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * TestSimulation.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the TestSimulation class
 *
 * Copyright (c) 2020 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "TestSimulation.h"
#include "../GridReference.h"
#include "../coordinateSystems/CartesianCoordinates2D.h"
#include "../coordinateSystems/GeographicalCoordinates2D.h"
//--------------------

// function definitions of the TestSimulation class
//-------------------------------------------------------------------

void TestSimulation::drawCreationOptions()
{
    ImGui::Checkbox("Random Vectors", &m_randomVectors);
    if(!m_randomVectors)
        ImGui::DragFloat2("Vector", &m_vectorValue.x);
}

std::shared_ptr<GridBase> TestSimulation::recreate(std::shared_ptr<CoordinateSystem> cs)
{
    m_cs = cs;
    m_grid = std::make_shared<TestSimGrid>(m_cs->getNumGridCells());

    // generate some data
    std::default_random_engine rng(mpu::getRanndomSeed());
    std::normal_distribution<float> dist(10,4);
    std::normal_distribution<float> vdist(0,4);

    for(int i : mpu::Range<int>(m_grid->size()))
    {
        float density = fmax(0,dist(rng));
        float velX = vdist(rng);
        float velY = vdist(rng);

        m_grid->write<AT::density>(i,density);
        if(m_randomVectors)
        {
            m_grid->write<AT::velocityX>(i, velX);
            m_grid->write<AT::velocityY>(i, velY);
        }
        else {
            m_grid->write<AT::velocityX>(i, m_vectorValue.x);
            m_grid->write<AT::velocityY>(i, m_vectorValue.y);
        }
    }

    // swap buffers and ready for rendering
    m_grid->swapAndRender();

    // select coordinate system
    switch(m_cs->getType())
    {
        case CSType::cartesian2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<CartesianCoordinates2D&>( *(this->m_cs)) ); };
            break;
        case CSType::geographical2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<GeographicalCoordinates2D&>( *(this->m_cs)) ); };
            break;
    }

    return m_grid;
}

std::unique_ptr<Simulation> TestSimulation::clone() const
{
    return std::make_unique<TestSimulation>(*this);
}

void TestSimulation::showGui(bool* show)
{
    ImGui::SetNextWindowSize({0,0},ImGuiCond_FirstUseEver);
    if(ImGui::Begin("RenderDemoSimulation",show))
    {
        std::string state;
        if(m_isPaused)
            ImGui::Text("State: Paused");
        else
            ImGui::Text("State: running");

        ImGui::Text("This is a rendering demo, so the simulation does nothing. There are also no settings.");
    }
    ImGui::End();
}

void TestSimulation::simulateOnce()
{
    m_simOnceFunc(); // calls correct template specialization
}

//CUDAHOSTDEV float centralDeriv(float left, float right, float delta)
//{
//    return (right-left) / 2.0f*delta;
//}

//CUDAHOSTDEV float2 gradient2D()

template <typename csT>
__global__ void testSimulation(TestSimGrid::ReferenceType grid, csT cs)
{
    for(int x : mpu::gridStrideRange( 1, cs.getNumGridCells3d().x-1 ))
        for(int y : mpu::gridStrideRangeY( 1, cs.getNumGridCells3d().y-1 ))
    {
        int3 cell{x,y,0};
        int cellId = cs.getCellId(cell);

        float rho = grid.read<AT::density>(cellId);
        float velX = grid.read<AT::velocityX>(cellId);
        float velY = grid.read<AT::velocityY>(cellId);

        grid.write<AT::velocityX>(cellId, velX);
        grid.write<AT::velocityY>(cellId, velY);
        grid.write<AT::density>(cellId,rho);

        // calculate gradient using central difference
        // since we use the density at at i and i+1 we get the gradient halfway in between the cells,
        // on the edge between cell i and i+1
        float rhoRight     = grid.read<AT::density>(cs.getRightNeighbor(cellId));
        float rhoForward   = grid.read<AT::density>(cs.getForwardNeighbor(cellId));

        float2 gradRho;
        gradRho.x = (rhoRight - rho) / cs.getCellSize().x;
        gradRho.y = (rhoForward - rho) / cs.getCellSize().y;

        grid.write<AT::densityGradX>(cellId, gradRho.x);
        grid.write<AT::densityGradY>(cellId, gradRho.y);

        // calculate divergence of the velocity field
        // remember, velocities are defined half way between the nodes,
        // we want the divergence at the node, so we get a central difference by looking at the velocities left and backwards from us
        // and compare them to our velocities
        float velLeftX = grid.read<AT::velocityX>(cs.getLeftNeighbor(cellId));
        float velBackwardY = grid.read<AT::velocityY>(cs.getBackwardNeighbor(cellId));

        float velDiv =  ((velX - velLeftX) / cs.getCellSize().x )
                      + ((velY - velBackwardY) / cs.getCellSize().x );

        grid.write<AT::velocityDiv>(cellId, velDiv);

        // laplace
        float rhoLeft     = grid.read<AT::density>(cs.getLeftNeighbor(cellId));
        float rhoBackward   = grid.read<AT::density>(cs.getBackwardNeighbor(cellId));

        float laplace =   (rhoRight - 2*rho + rhoLeft) / (cs.getCellSize().x*cs.getCellSize().x)
                        + ( (rhoForward - 2*rho + rhoBackward) / (cs.getCellSize().y*cs.getCellSize().y) );

        grid.write<AT::densityLaplace>(cellId, laplace*0.001);

        // curl is more difficult, as we can only compute it at cell corners
        // offsetted from where we want to visualize it
        // so we need to compute 4 curls and average them

        // forward right quadrant
        float velRightY = grid.read<AT::velocityY>(cs.getRightNeighbor(cellId));
        float velForwardX = grid.read<AT::velocityX>(cs.getForwardNeighbor(cellId));

        float forwardRightCurl = ( (velRightY-velY) / cs.getCellSize().x )
                                -( (velForwardX-velX) / cs.getCellSize().y );

        grid.write<AT::velocityCurl>(cellId, forwardRightCurl);
    }
}

template <typename csT>
__global__ void interpolateCurl(TestSimGrid::ReferenceType grid, csT cs)
{
    for(int x : mpu::gridStrideRange( 1, cs.getNumGridCells3d().x-1 ))
        for(int y : mpu::gridStrideRangeY( 1, cs.getNumGridCells3d().y-1 ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);

            // only forward right curl was computed above, so now curl must be interpolated
            float curlForwardRight = grid.read<AT::velocityCurl>(cellId);
            float curlForwardLeft = grid.read<AT::velocityCurl>(cs.getLeftNeighbor(cellId));
            float curlBackwardsRight = grid.read<AT::velocityCurl>(cs.getBackwardNeighbor(cellId));
            float curlBackwardsLeft = grid.read<AT::velocityCurl>(cs.getLeftNeighbor(cs.getBackwardNeighbor(cellId)));

            float averageCurl = curlForwardRight + curlForwardLeft + curlBackwardsRight + curlBackwardsLeft;
            averageCurl *= 0.25;

            grid.write<AT::velocityCurl>(cellId, averageCurl);

            // copy all other values to the new buffer
            grid.copy<AT::velocityX>(cellId);
            grid.copy<AT::velocityY>(cellId);
            grid.copy<AT::density>(cellId);
            grid.copy<AT::densityGradX>(cellId);
            grid.copy<AT::densityGradY>(cellId);
            grid.copy<AT::densityLaplace>(cellId);
            grid.copy<AT::velocityDiv>(cellId);
        }
}

template <typename csT>
void TestSimulation::simulateOnceImpl(csT& cs)
{
    dim3 blocksize{16,16,1};
    dim3 numBlocks{ static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().x ,blocksize.x)),
                    static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().y ,blocksize.y)), 1};

    testSimulation<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs);
    m_grid->swapBuffer();
    interpolateCurl<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs);
}

template void TestSimulation::simulateOnceImpl<CartesianCoordinates2D>(CartesianCoordinates2D& cs);

GridBase& TestSimulation::getGrid()
{
    return *m_grid;
}
