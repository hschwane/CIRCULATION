#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * TestSimulation.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the TestSimulation class
 *
 * Copyright (c) 2020 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "TestSimulation.h"
#include "../GridReference.h"
#include "../coordinateSystems/CartesianCoordinates2D.h"
#include "../coordinateSystems/GeographicalCoordinates2D.h"
//--------------------

// function definitions of the TestSimulation class
//-------------------------------------------------------------------

void TestSimulation::drawCreationOptions()
{
    ImGui::Checkbox("Random Vectors", &m_randomVectors);
    if(!m_randomVectors)
        ImGui::DragFloat2("Vector", &m_vectorValue.x);
}

std::shared_ptr<GridBase> TestSimulation::recreate(std::shared_ptr<CoordinateSystem> cs)
{
    m_cs = cs;
    m_grid = std::make_shared<TestSimGrid>(m_cs->getNumGridCells());

    // generate some data
    std::default_random_engine rng(mpu::getRanndomSeed());
    std::normal_distribution<float> dist(10,4);
    std::normal_distribution<float> vdist(0,4);

    for(int i : mpu::Range<int>(m_grid->size()))
    {
        float density = fmax(0,dist(rng));
        float velX = vdist(rng);
        float velY = vdist(rng);

        m_grid->write<AT::density>(i,density);
        if(m_randomVectors)
        {
            m_grid->write<AT::velocityX>(i, velX);
            m_grid->write<AT::velocityY>(i, velY);
        }
        else {
            m_grid->write<AT::velocityX>(i, m_vectorValue.x);
            m_grid->write<AT::velocityY>(i, m_vectorValue.y);
        }
    }

    // swap buffers and ready for rendering
    m_grid->swapAndRender();

    // select coordinate system
    switch(m_cs->getType())
    {
        case CSType::cartesian2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<CartesianCoordinates2D&>( *(this->m_cs)) ); };
            break;
        case CSType::geographical2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<GeographicalCoordinates2D&>( *(this->m_cs)) ); };
            break;
    }

    return m_grid;
}

std::unique_ptr<Simulation> TestSimulation::clone() const
{
    return std::make_unique<TestSimulation>(*this);
}

void TestSimulation::showGui(bool* show)
{
    ImGui::SetNextWindowSize({0,0},ImGuiCond_FirstUseEver);
    if(ImGui::Begin("RenderDemoSimulation",show))
    {
        std::string state;
        if(m_isPaused)
            ImGui::Text("State: Paused");
        else
            ImGui::Text("State: running");

        ImGui::Text("This is a rendering demo, so the simulation does nothing. There are also no settings.");
    }
    ImGui::End();
}

void TestSimulation::simulateOnce()
{
    m_simOnceFunc(); // calls correct template specialization
}

//CUDAHOSTDEV float centralDeriv(float left, float right, float delta)
//{
//    return (right-left) / 2.0f*delta;
//}

//CUDAHOSTDEV float2 gradient2D()

template <typename csT>
__global__ void testSimulation(TestSimGrid::ReferenceType grid, csT cs)
{
    for(int x : mpu::gridStrideRange( 1, cs.getNumGridCells3d().x-1 ))
        for(int y : mpu::gridStrideRangeY( 1, cs.getNumGridCells3d().y-1 ))
    {
        int3 cell{x,y,0};
        int cellId = cs.getCellId(cell);

        float rho = grid.read<AT::density>(cellId);
        float velx = grid.read<AT::velocityX>(cellId);
        float vely = grid.read<AT::velocityY>(cellId);

        grid.write<AT::velocityX>(cellId, velx);
        grid.write<AT::velocityY>(cellId, vely);
        grid.write<AT::density>(cellId,rho);

        // calculate gradient using central difference
        // since we use the density at at i and i+1 we get the gradient halfway in between the cells,
        // on the edge between cell i and i+1
        float rhoRight     = grid.read<AT::density>(cs.getRightNeighbor(cellId));
        float rhoForward   = grid.read<AT::density>(cs.getForwardNeighbor(cellId));

        float2 gradRho;
        gradRho.x = (rhoRight - rho) / cs.getCellSize().x;
        gradRho.y = (rhoForward - rho) / cs.getCellSize().y;

        grid.write<AT::densityGradX>(cellId, gradRho.x);
        grid.write<AT::densityGradY>(cellId, gradRho.y);

        // calculate divergence of the velocity field
        // remember, velocities are defined half way between the nodes,
        // we want the divergence at the node, so we get a central difference by looking at the velocities left and backwards from us
        // and compare them to our velocities
        float velLeft = grid.read<AT::velocityX>(cs.getLeftNeighbor(cellId));
        float velBackward = grid.read<AT::velocityX>(cs.getBackwardNeighbor(cellId));

        float velDiv =  ( (velx-velLeft) / cs.getCellSize().x )
                      + ( (vely-velBackward) / cs.getCellSize().x );

        grid.write<AT::velocityDiv>(cellId, velDiv);
    }
}

template <typename csT>
void TestSimulation::simulateOnceImpl(csT& cs)
{
    dim3 blocksize{16,16,1};
    dim3 numBlocks{ static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().x ,blocksize.x)),
                    static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().y ,blocksize.y)), 1};

    testSimulation<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs);
}

template void TestSimulation::simulateOnceImpl<CartesianCoordinates2D>(CartesianCoordinates2D& cs);

GridBase& TestSimulation::getGrid()
{
    return *m_grid;
}
