#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * TestSimulation.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the TestSimulation class
 *
 * Copyright (c) 2020 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "TestSimulation.h"
#include "../GridReference.h"
#include "../coordinateSystems/CartesianCoordinates2D.h"
#include "../coordinateSystems/GeographicalCoordinates2D.h"
#include "../finiteDifferences.h"
#include "../boundaryConditions.h"
//--------------------

// function definitions of the TestSimulation class
//-------------------------------------------------------------------

void TestSimulation::showCreationOptions()
{
    ImGui::Checkbox("Random velocity vectors", &m_randomVectors);
    if(!m_randomVectors)
        ImGui::DragFloat2("Vector", &m_vectorValue.x);
}

void TestSimulation::showBoundaryOptions(const CoordinateSystem& cs)
{
    if(cs.hasBoundary().x)
    {
        ImGui::Text("X-Axis Boundary:");
        if(ImGui::RadioButton("Isolated##X",m_boundaryIsolatedX))
            m_boundaryIsolatedX = true;

        ImGui::SameLine();
        if(ImGui::RadioButton("Const. temperature##X", !m_boundaryIsolatedX))
        {
            m_boundaryIsolatedX = false;
            m_needUpdateBoundaries = true;
        }

        if(!m_boundaryIsolatedX)
            if(ImGui::DragFloat("Temperature on boundary##X", &m_boundaryTemperatureX, 0.1))
                m_needUpdateBoundaries = true;
    }

    if(cs.hasBoundary().y)
    {
        ImGui::Text("Y-Axis Boundary:");
        if(ImGui::RadioButton("Isolated##Y",m_boundaryIsolatedY))
            m_boundaryIsolatedY = true;

        ImGui::SameLine();
        if(ImGui::RadioButton("Const. temperature##Y", !m_boundaryIsolatedY))
        {
            m_boundaryIsolatedY = false;
            m_needUpdateBoundaries = true;
        }

        if(!m_boundaryIsolatedY)
            if(ImGui::DragFloat("Temperature on boundary##Y", &m_boundaryTemperatureY, 0.1))
                m_needUpdateBoundaries = true;
    }
}

void TestSimulation::showSimulationOptions()
{
    ImGui::Checkbox("diffuse heat",&m_diffuseHeat);
    ImGui::Checkbox("use divergence of gradient instead of laplacian",&m_useDivOfGrad);
    ImGui::Checkbox("use leapfrog (unstable)",&m_leapfrogIntegrattion);
    ImGui::Checkbox("advect heat",&m_advectHeat);
    ImGui::DragFloat("Heat Coefficient",&m_heatCoefficient,0.0001,0.0001f,1.0,"%.4f");
    ImGui::DragFloat("Timestep",&m_timestep,0.0001,0.0001f,1.0,"%.4f");
    ImGui::Text("Biggest maybe stable timestep is %f.",
                (fmin(m_cs->getCellSize().x,m_cs->getCellSize().y) * fmin(m_cs->getCellSize().x,m_cs->getCellSize().y) / (2*m_heatCoefficient) ) );
    ImGui::Text("Simulated Time units: %f", m_totalSimulatedTime);

    if( ImGui::CollapsingHeader("Boundaries"))
        showBoundaryOptions(*m_cs);
}

std::shared_ptr<GridBase> TestSimulation::recreate(std::shared_ptr<CoordinateSystem> cs)
{
    m_cs = cs;
    m_grid = std::make_shared<TestSimGrid>(m_cs->getNumGridCells());
    m_offsettedCurl.resize(m_cs->getNumGridCells());

    // select coordinate system
    switch(m_cs->getType())
    {
        case CSType::cartesian2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<CartesianCoordinates2D&>( *(this->m_cs)) ); };
            break;
        case CSType::geographical2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<GeographicalCoordinates2D&>( *(this->m_cs)) ); };
            break;
    }

    reset();
    return m_grid;
}

void TestSimulation::reset()
{
    // generate some data
    std::default_random_engine rng(mpu::getRanndomSeed());
    std::normal_distribution<float> dist(10,4);
    std::normal_distribution<float> vdist(0,4);

    m_grid->cacheOverwrite();
    for(int i : mpu::Range<int>(m_grid->size()))
    {
        float density = fmax(0,dist(rng));
        float temperature = fmax(0,dist(rng));
        float velX = vdist(rng);
        float velY = vdist(rng);

        m_grid->initialize<AT::density>(i,density);
        m_grid->initialize<AT::temperature>(i,temperature);
        if(m_randomVectors)
        {
            m_grid->initialize<AT::velocityX>(i, velX);
            m_grid->initialize<AT::velocityY>(i, velY);
        }
        else {
            m_grid->initialize<AT::velocityX>(i, m_vectorValue.x);
            m_grid->initialize<AT::velocityY>(i, m_vectorValue.y);
        }
    }

    // initialize boundary
    initializeFixedValueBoundaries<AT::temperature>(!m_boundaryIsolatedX && m_cs->hasBoundary().x,
                                                    !m_boundaryIsolatedY && m_cs->hasBoundary().y,
                                                    m_boundaryTemperatureX, m_boundaryTemperatureY, *m_cs, *m_grid);

    // swap buffers and ready for rendering
    m_grid->pushCachToDevice();
    m_grid->swapAndRender();

    // reset simulation state
    m_totalSimulatedTime = 0;
    m_firstTimestep = true;
    m_needUpdateBoundaries = false;
}

std::unique_ptr<Simulation> TestSimulation::clone() const
{
    return std::make_unique<TestSimulation>(*this);
}

void TestSimulation::simulateOnce()
{
    m_simOnceFunc(); // calls correct template specialization
}

template <typename csT>
__global__ void testSimulationA(TestSimGrid::ReferenceType grid, csT coordinateSystem, mpu::VectorReference<float> offsettedCurl,
        bool diffuseHeat, bool advectHeat, float heatCoefficient, bool useDivOfGrad, float timestep)
{
    csT cs = coordinateSystem;

    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-cs.hasBoundary().y ))
    {
        int3 cell{x,y,0};
        int cellId = cs.getCellId(cell);
        float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

        // do bounds checking
        int3 leftNeigbour = cs.getCellId3d(cs.getRightNeighbor(cellId));
        int3 rightNeibor = cs.getCellId3d(cs.getLeftNeighbor(cellId));
        int3 backwardNeigbor = cs.getCellId3d(cs.getBackwardNeighbor(cellId));
        int3 forwardNeigbor = cs.getCellId3d(cs.getForwardNeighbor(cellId));

        auto oob = [&](int3 c)->bool
        {
            return (c.x >= cs.getNumGridCells3d().x) || (c.x < 0) || (c.y >= cs.getNumGridCells3d().y) || (c.y < 0);
        };

        if(oob(leftNeigbour))
            printf("Left neighbor out of bounds! cell (%i,%i) \n",x,y);
        if(oob(rightNeibor))
            printf("Right neighbor out of bounds! cell (%i,%i) \n",x,y);
        if(oob(backwardNeigbor))
            printf("Backward neighbor out of bounds! cell (%i,%i) \n",x,y);
        if(oob(forwardNeigbor))
            printf("Forward neighbor out of bounds! cell (%i,%i) \n",x,y);

        float rho = grid.read<AT::density>(cellId);
        float velX = grid.read<AT::velocityX>(cellId);
        float velY = grid.read<AT::velocityY>(cellId);

        // calculate gradient using central difference
        // since we use the density at at i and i+1 we get the gradient halfway in between the cells,
        // on the edge between cell i and i+1
        float rhoRight     = grid.read<AT::density>(cs.getRightNeighbor(cellId));
        float rhoForward   = grid.read<AT::density>(cs.getForwardNeighbor(cellId));

        float2 gradRho = gradient2d(rho, rhoRight, rho, rhoForward, cellPos, cs);

        grid.write<AT::densityGradX>(cellId, gradRho.x);
        grid.write<AT::densityGradY>(cellId, gradRho.y);

        // calculate divergence of the velocity field
        // remember, velocities are defined half way between the nodes,
        // we want the divergence at the node, so we get a central difference by looking at the velocities left and backwards from us
        // and compare them to our velocities
        float velLeftX = grid.read<AT::velocityX>(cs.getLeftNeighbor(cellId));
        float velBackwardY = grid.read<AT::velocityY>(cs.getBackwardNeighbor(cellId));

        float velDiv = divergence2d(velLeftX,velX,velBackwardY,velY,cellPos,cs);

        grid.write<AT::velocityDiv>(cellId, velDiv);

        // laplace
        float rhoLeft     = grid.read<AT::density>(cs.getLeftNeighbor(cellId));
        float rhoBackward   = grid.read<AT::density>(cs.getBackwardNeighbor(cellId));

        float laplace = laplace2d(rhoLeft,rhoRight,rhoBackward,rhoForward,rho,cellPos,cs);

        grid.write<AT::densityLaplace>(cellId, laplace);

        // curl is more difficult, as we can only compute it at cell corners
        // offsetted from where we want to visualize it
        // so we need to compute 4 curls and average them

        // forward right quadrant
        float velRightY = grid.read<AT::velocityY>(cs.getRightNeighbor(cellId));
        float velForwardX = grid.read<AT::velocityX>(cs.getForwardNeighbor(cellId));

        float forwardRightCurl = curl2d(velY,velRightY, velX, velForwardX,cellPos,cs);
        // averaging is done in the next kernel
        offsettedCurl[cellId] = forwardRightCurl;

        // temperature gradient

        float temp = grid.read<AT::temperature>(cellId);
        float tempRight = grid.read<AT::temperature>(cs.getRightNeighbor(cellId));
        float tempForward = grid.read<AT::temperature>(cs.getForwardNeighbor(cellId));

        float2 tempGrad = gradient2d(temp,tempRight,temp,tempForward,cellPos,cs);

        grid.write<AT::temperatureGradX>(cellId,tempGrad.x);
        grid.write<AT::temperatureGradY>(cellId,tempGrad.y);
    }
}

template <typename csT>
__global__ void testSimulationB(TestSimGrid::ReferenceType grid, csT coordinateSystem, mpu::VectorReference<const float> offsettedCurl,
                                bool useLeapfrog, bool diffuseHeat, bool advectHeat, float heatCoefficient, bool useDivOfGrad, float timestep)
{
    csT cs = coordinateSystem;

    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-cs.hasBoundary().y ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);
            float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

            // only forward right curl was computed above, so now curl must be interpolated
            float curlForwardRight = offsettedCurl[cellId];
            float curlForwardLeft = offsettedCurl[cs.getLeftNeighbor(cellId)];
            float curlBackwardsRight = offsettedCurl[cs.getBackwardNeighbor(cellId)];
            float curlBackwardsLeft = offsettedCurl[cs.getLeftNeighbor(cs.getBackwardNeighbor(cellId))];

            float averageCurl = curlForwardRight + curlForwardLeft + curlBackwardsRight + curlBackwardsLeft;
            averageCurl *= 0.25;

            grid.write<AT::velocityCurl>(cellId, averageCurl);

            // solve the heat equation
            if(diffuseHeat || advectHeat)
            {
                float temp_dt =0;
                float temp = grid.read<AT::temperature>(cellId);

                if(diffuseHeat)
                {
                    float tempGradX = grid.readNext<AT::temperatureGradX>(cellId);
                    float tempGradY = grid.readNext<AT::temperatureGradY>(cellId);
                    float tempGradXLeft = grid.readNext<AT::temperatureGradX>(cs.getLeftNeighbor(cellId));
                    float tempGradYBack = grid.readNext<AT::temperatureGradY>(cs.getBackwardNeighbor(cellId));

                    float heatDivGrad = divergence2d(tempGradXLeft, tempGradX, tempGradYBack, tempGradY, cellPos, cs);

                    float tempLeft = grid.read<AT::temperature>(cs.getLeftNeighbor(cellId));
                    float tempRight = grid.read<AT::temperature>(cs.getRightNeighbor(cellId));
                    float tempForward = grid.read<AT::temperature>(cs.getForwardNeighbor(cellId));
                    float tempBackward = grid.read<AT::temperature>(cs.getBackwardNeighbor(cellId));

                    float heatLaplace = laplace2d(tempLeft,tempRight,tempBackward,tempForward,temp,cellPos,cs);

                    if(useDivOfGrad)
                        temp_dt += heatCoefficient * heatDivGrad;
                    else
                        temp_dt += heatCoefficient *heatLaplace;
                }

                if(advectHeat)
                {
                    temp_dt -= grid.readNext<AT::velocityDiv>(cellId) * temp;
                }

                float previousTemp;
                if(useLeapfrog)
                {
                    previousTemp = grid.readPrev<AT::temperature>(cellId);
                    timestep *=2.0f;
                }
                else
                {
                    previousTemp = temp;
                }

                float nextTemp =  previousTemp + temp_dt * timestep;
                grid.write<AT::temperature>(cellId,nextTemp);
            }
            else
                grid.copy<AT::temperature>(cellId);
        }
}

template <typename csT>
void TestSimulation::simulateOnceImpl(csT& cs)
{
    dim3 blocksize{16,16,1};
    dim3 numBlocks{ static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().x ,blocksize.x)),
                    static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().y ,blocksize.y)), 1};

    if(m_needUpdateBoundaries)
    {
//        m_grid->cacheOnHost();
        initializeFixedValueBoundaries<AT::temperature>(!m_boundaryIsolatedX && m_cs->hasBoundary().x,
                                                        !m_boundaryIsolatedY && m_cs->hasBoundary().y,
                                                        m_boundaryTemperatureX, m_boundaryTemperatureY, *m_cs, *m_grid);
//        m_grid->pushCachToDevice();
    }

    handleMirroredBoundaries<AT::temperature>(m_boundaryIsolatedX && cs.hasBoundary().x,
                                              m_boundaryIsolatedY && cs.hasBoundary().y,
                                              cs, *m_grid);

    if(m_diffuseHeat)
        m_totalSimulatedTime += m_timestep;

    testSimulationA<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs,m_offsettedCurl.getVectorReference(),
            m_diffuseHeat,m_advectHeat,m_heatCoefficient,m_useDivOfGrad,m_timestep);
    testSimulationB<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs,m_offsettedCurl.getVectorReference(),
            !m_firstTimestep && m_leapfrogIntegrattion,m_diffuseHeat,m_advectHeat,m_heatCoefficient,m_useDivOfGrad,m_timestep);

    m_firstTimestep = false;
}

GridBase& TestSimulation::getGrid()
{
    return *m_grid;
}

std::string TestSimulation::getDisplayName()
{
    return "Test Simulation";
}
