#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * TestSimulation.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the TestSimulation class
 *
 * Copyright (c) 2020 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "TestSimulation.h"
#include "../GridReference.h"
#include "../coordinateSystems/CartesianCoordinates2D.h"
#include "../coordinateSystems/GeographicalCoordinates2D.h"
#include "../finiteDifferences.h"
//--------------------

// function definitions of the TestSimulation class
//-------------------------------------------------------------------

void TestSimulation::drawCreationOptions()
{
    ImGui::Checkbox("Random Vectors", &m_randomVectors);
    if(!m_randomVectors)
        ImGui::DragFloat2("Vector", &m_vectorValue.x);

    ImGui::DragFloat("Boundary Temperature", &m_boundaryTemperature);
}

std::shared_ptr<GridBase> TestSimulation::recreate(std::shared_ptr<CoordinateSystem> cs)
{
    m_cs = cs;
    m_grid = std::make_shared<TestSimGrid>(m_cs->getNumGridCells());

    // generate some data
    std::default_random_engine rng(mpu::getRanndomSeed());
    std::normal_distribution<float> dist(10,4);
    std::normal_distribution<float> vdist(0,4);

    for(int i : mpu::Range<int>(m_grid->size()))
    {
        float density = fmax(0,dist(rng));
        float temperature = fmax(0,dist(rng));
        float velX = vdist(rng);
        float velY = vdist(rng);

        m_grid->write<AT::density>(i,density);
        m_grid->write<AT::temperature>(i,temperature);
        if(m_randomVectors)
        {
            m_grid->write<AT::velocityX>(i, velX);
            m_grid->write<AT::velocityY>(i, velY);
        }
        else {
            m_grid->write<AT::velocityX>(i, m_vectorValue.x);
            m_grid->write<AT::velocityY>(i, m_vectorValue.y);
        }
    }

    // initialize boundary

    // calculate number of boundary cells
    int numBoundCells = 2* m_cs->hasBoundary().y * m_cs->getNumGridCells3d().x ;
    for(int i : mpu::Range<int>(numBoundCells))
    {
        // transform boundary cell id into actual cell id
        int cellId = i;
        if(cellId >= m_cs->getNumGridCells3d().x)
            cellId += m_cs->getNumGridCells3d().x * (m_cs->getNumGridCells3d().y-2);

        m_grid->initialize<AT::temperature>(cellId,m_boundaryTemperature);
    }

    // swap buffers and ready for rendering
    m_grid->swapAndRender();

    // select coordinate system
    switch(m_cs->getType())
    {
        case CSType::cartesian2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<CartesianCoordinates2D&>( *(this->m_cs)) ); };
            break;
        case CSType::geographical2d:
            m_simOnceFunc = [this](){ this->simulateOnceImpl( static_cast<GeographicalCoordinates2D&>( *(this->m_cs)) ); };
            break;
    }

    m_totalSimulatedTime = 0;
    return m_grid;
}

std::unique_ptr<Simulation> TestSimulation::clone() const
{
    return std::make_unique<TestSimulation>(*this);
}

void TestSimulation::showGui(bool* show)
{
    ImGui::SetNextWindowSize({0,0},ImGuiCond_FirstUseEver);
    if(ImGui::Begin("Test Simulation",show))
    {
        if(m_isPaused)
        {
            ImGui::Text("State: Paused");
            if(ImGui::Button("Resume")) resume();
        }
        else
        {
            ImGui::Text("State: running");
            if(ImGui::Button("Pause")) pause();
        }

        ImGui::Checkbox("diffuse heat",&m_diffuseHeat);
        ImGui::Checkbox("advect heat",&m_advectHeat);
        ImGui::DragFloat("Heat Coefficient",&m_heatCoefficient,0.01);
        ImGui::DragFloat("Timestep",&m_timestep,0.01);
        ImGui::Text("Simulated Time units: %f", m_totalSimulatedTime);
    }
    ImGui::End();
}

void TestSimulation::simulateOnce()
{
    m_simOnceFunc(); // calls correct template specialization
}

template <typename csT>
__global__ void testSimulation(TestSimGrid::ReferenceType grid, csT coordinateSystem, bool diffuseHeat, bool advectHeat, float heatCoefficient, float timestep)
{
    csT cs = coordinateSystem;

    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-cs.hasBoundary().y ))
    {
        int3 cell{x,y,0};
        int cellId = cs.getCellId(cell);
        float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

        // do bounds checking
        int3 leftNeigbour = cs.getCellId3d(cs.getRightNeighbor(cellId));
        int3 rightNeibor = cs.getCellId3d(cs.getLeftNeighbor(cellId));
        int3 backwardNeigbor = cs.getCellId3d(cs.getBackwardNeighbor(cellId));
        int3 forwardNeigbor = cs.getCellId3d(cs.getForwardNeighbor(cellId));

        auto oob = [&](int3 c)->bool
        {
            return (c.x >= cs.getNumGridCells3d().x) || (c.x < 0) || (c.y >= cs.getNumGridCells3d().y) || (c.y < 0);
        };

        if(oob(leftNeigbour))
            printf("Left neighbor out of bounds! cell (%i,%i) \n",x,y);
        if(oob(rightNeibor))
            printf("Right neighbor out of bounds! cell (%i,%i) \n",x,y);
        if(oob(backwardNeigbor))
            printf("Backward neighbor out of bounds! cell (%i,%i) \n",x,y);
        if(oob(forwardNeigbor))
            printf("Forward neighbor out of bounds! cell (%i,%i) \n",x,y);

        float rho = grid.read<AT::density>(cellId);
        float velX = grid.read<AT::velocityX>(cellId);
        float velY = grid.read<AT::velocityY>(cellId);

        grid.write<AT::velocityX>(cellId, velX);
        grid.write<AT::velocityY>(cellId, velY);
        grid.write<AT::density>(cellId,rho);

        // calculate gradient using central difference
        // since we use the density at at i and i+1 we get the gradient halfway in between the cells,
        // on the edge between cell i and i+1
        float rhoRight     = grid.read<AT::density>(cs.getRightNeighbor(cellId));
        float rhoForward   = grid.read<AT::density>(cs.getForwardNeighbor(cellId));

        float2 gradRho = gradient2d(rho, rhoRight, rho, rhoForward, cellPos, cs);

        grid.write<AT::densityGradX>(cellId, gradRho.x);
        grid.write<AT::densityGradY>(cellId, gradRho.y);

        // calculate divergence of the velocity field
        // remember, velocities are defined half way between the nodes,
        // we want the divergence at the node, so we get a central difference by looking at the velocities left and backwards from us
        // and compare them to our velocities
        float velLeftX = grid.read<AT::velocityX>(cs.getLeftNeighbor(cellId));
        float velBackwardY = grid.read<AT::velocityY>(cs.getBackwardNeighbor(cellId));

        float velDiv = divergence2d(velLeftX,velX,velBackwardY,velY,cellPos,cs);

        grid.write<AT::velocityDiv>(cellId, velDiv);

        // laplace
        float rhoLeft     = grid.read<AT::density>(cs.getLeftNeighbor(cellId));
        float rhoBackward   = grid.read<AT::density>(cs.getBackwardNeighbor(cellId));

        float laplace = laplace2d(rhoLeft,rhoRight,rhoBackward,rhoForward,rho,cellPos,cs);

        grid.write<AT::densityLaplace>(cellId, laplace);

        // curl is more difficult, as we can only compute it at cell corners
        // offsetted from where we want to visualize it
        // so we need to compute 4 curls and average them

        // forward right quadrant
        float velRightY = grid.read<AT::velocityY>(cs.getRightNeighbor(cellId));
        float velForwardX = grid.read<AT::velocityX>(cs.getForwardNeighbor(cellId));

        float forwardRightCurl = curl2d(velY,velRightY, velX, velForwardX,cellPos,cs);
        // averaging is done in the next kernel
        grid.write<AT::velocityCurl>(cellId, forwardRightCurl);

        // solve the heat equation
        if(diffuseHeat || advectHeat)
        {
            float temp_dt =0;
            float temp = grid.read<AT::temperature>(cellId);

            if(diffuseHeat)
            {
                float tempLeft = grid.read<AT::temperature>(cs.getLeftNeighbor(cellId));
                float tempRight = grid.read<AT::temperature>(cs.getRightNeighbor(cellId));
                float tempForward = grid.read<AT::temperature>(cs.getForwardNeighbor(cellId));
                float tempBackward = grid.read<AT::temperature>(cs.getBackwardNeighbor(cellId));

                float heatLaplace = laplace2d(tempLeft, tempRight, tempBackward, tempForward, temp,cellPos,cs);
                temp_dt += heatCoefficient *heatLaplace;
            }

            if(advectHeat)
            {
                temp_dt -= velDiv * temp;
            }

            temp += temp_dt * timestep;
            grid.write<AT::temperature>(cellId,temp);
        }
        else
            grid.copy<AT::temperature>(cellId);
    }
}

template <typename csT>
__global__ void interpolateCurl(TestSimGrid::ReferenceType grid, csT cs)
{
    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-cs.hasBoundary().y ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);

            // only forward right curl was computed above, so now curl must be interpolated
            float curlForwardRight = grid.read<AT::velocityCurl>(cellId);
            float curlForwardLeft = grid.read<AT::velocityCurl>(cs.getLeftNeighbor(cellId));
            float curlBackwardsRight = grid.read<AT::velocityCurl>(cs.getBackwardNeighbor(cellId));
            float curlBackwardsLeft = grid.read<AT::velocityCurl>(cs.getLeftNeighbor(cs.getBackwardNeighbor(cellId)));

            float averageCurl = curlForwardRight + curlForwardLeft + curlBackwardsRight + curlBackwardsLeft;
            averageCurl *= 0.25;

            grid.write<AT::velocityCurl>(cellId, averageCurl);

            // copy all other values to the new buffer
            grid.copy<AT::velocityX>(cellId);
            grid.copy<AT::velocityY>(cellId);
            grid.copy<AT::density>(cellId);
            grid.copy<AT::densityGradX>(cellId);
            grid.copy<AT::densityGradY>(cellId);
            grid.copy<AT::densityLaplace>(cellId);
            grid.copy<AT::velocityDiv>(cellId);
            grid.copy<AT::temperature>(cellId);
        }
}

template <typename csT>
void TestSimulation::simulateOnceImpl(csT& cs)
{
    dim3 blocksize{16,16,1};
    dim3 numBlocks{ static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().x ,blocksize.x)),
                    static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().y ,blocksize.y)), 1};

    if(m_diffuseHeat)
        m_totalSimulatedTime += m_timestep;
    testSimulation<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs,m_diffuseHeat,m_advectHeat,m_heatCoefficient,m_timestep);
    m_grid->swapBuffer();
    interpolateCurl<<< numBlocks, blocksize>>>(m_grid->getGridReference(),cs);
}

template void TestSimulation::simulateOnceImpl<CartesianCoordinates2D>(CartesianCoordinates2D& cs);

GridBase& TestSimulation::getGrid()
{
    return *m_grid;
}
