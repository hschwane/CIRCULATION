#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * ShallowWaterModel.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the ShallowWaterModel class
 *
 * Copyright (c) 2020 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "CosineAdvection.h"

#include <mpUtils/mpUtils.h>
#include <mpUtils/mpGraphics.h>
#include <mpUtils/mpCuda.h>

#include "../GridReference.h"
#include "../finiteDifferences.h"
#include "../boundaryConditions.h"
//--------------------

// function definitions of the ShallowWaterModel class
//-------------------------------------------------------------------

void CosineAdvection::showCreationOptions()
{
    ImGui::Text("Test Case number 1 from David L. Williamson 1992.");
    ImGui::DragFloat("Wind Angle offset (alpha) in rad", &m_alpha, 0.001f,0.0,M_PI_2);
    ImGui::DragFloat("Wind Velocity (u0) in m/s", &m_u0SI, 0.001f);
    ImGui::DragFloat("Earth radius (a) in m", &m_earthRadiusSI);
    ImGui::DragFloat("Angular Velocity in rad/m", &m_angularVelocitySI, 0.00001f, 0.00001f, 5.0f, "%.8f");
    ImGui::DragFloat2("position of cosine bell", &m_cosineBellCenter.x, 0.001);
    ImGui::DragFloat("cosine bell radius (R) in m", &m_cosineBellRadiusSI, 1.0f);
    ImGui::DragFloat("Internal time unit in s", &m_timeUnit, 0.1f, 1.0);
}

void CosineAdvection::showBoundaryOptions(const CoordinateSystem& cs)
{
}

void CosineAdvection::showSimulationOptions()
{
    ImGui::DragFloat("Angular Velocity", &m_angularVelocitySI, 0.00001f, 0.00001, 5.0f, "%.5f");

    ImGui::DragFloat("Geopotential diffusion",&m_geopotDiffusion,0.00001f,0.00001,1.0,"%.5f");
    ImGui::Checkbox("Use Leapfrog",&m_useLeapfrog);
    ImGui::DragFloat("Timestep",&m_timestep,0.000001,0.000001f,1.0,"%.6f");
    ImGui::Text("Simulated Time units: %f", m_totalSimulatedTime);
}

std::shared_ptr<GridBase> CosineAdvection::recreate(std::shared_ptr<CoordinateSystem> cs)
{
    m_cs = cs;
    m_grid = std::make_shared<ShallowWaterGrid>(m_cs->getNumGridCells());
    m_phiPlusKBuffer.resize(m_cs->getNumGridCells());
    m_vortPlusCor.resize(m_cs->getNumGridCells());

    if(m_cs->getType() != CSType::geographical2d)
    {
        logERROR("CosineAdvection") << "Pole advection test loaded in cartesian cordinates!";
        tinyfd_messageBox("Error","Advection Test only works for geographical coordinates",
                          "ok", "error",1);
        return m_grid;
    }

    // scale units
    m_lengthUnit = m_earthRadiusSI / m_cs->getMinCoord().z;
    logINFO("PoleAdvectionTest") << "Internal length unit: " << m_lengthUnit << " meter";
    logINFO("PoleAdvectionTest") << "Internal time unit: " << m_timeUnit << " seconds";

    m_earthRadius = m_earthRadiusSI / m_lengthUnit;
    m_u0 = m_u0SI / m_lengthUnit * m_timeUnit;
    m_angularVelocity = m_angularVelocitySI * m_timeUnit;
    m_cosineBellRadius = m_cosineBellRadiusSI / m_lengthUnit;
    m_h0 = m_h0SI / m_lengthUnit;
    m_g = m_gSI / m_lengthUnit * m_timeUnit * m_timeUnit;

    logINFO("PoleAdvectionTest") << "Settings in internal units: earth radius: " << m_earthRadius << ", u0: " << m_u0
                                 << ", angular velocity: " << m_angularVelocity
                                 << ", cosine bell radius: " << m_cosineBellRadius
                                 << ", cosine bell heigt: " << m_h0
                                 << ", g: " << m_g
                                 ;

    reset();
    return m_grid;
}

void CosineAdvection::reset()
{
    m_grid->cacheOverwrite();

    float cosAlpha = cos(m_alpha);
    float sinAlpha = sin(m_alpha);

    float sinLatCenter = sin(m_cosineBellCenter.y);
    float cosLatCenter = cos(m_cosineBellCenter.y);

    // create initial conditions using gaussian
    for(int i : mpu::Range<int>(m_grid->size()))
    {
        float3 c = m_cs->getCellCoordinate(i);

        float sinLat = sin(c.y);
        float cosLat = cos(c.y);

        float velX = m_u0*(cosLat*cosAlpha + sinLat*cos(c.x)*sinAlpha);
        float velY = -m_u0*sinLat*sinAlpha;

        float geopotential = 0;
        float r = m_earthRadius * acos( sinLatCenter*sinLat + cosLatCenter*cosLat*cos(c.x - m_cosineBellCenter.x));
        if(r < m_cosineBellRadius)
        {
            float h = (m_h0/2.0f) * (1.0f + cos( M_PI * r / m_cosineBellRadius ));
            geopotential = m_g*h;
        }

        m_grid->initialize<AT::geopotential>(i, geopotential);
        m_grid->initialize<AT::velocityX>(i, velX);
        m_grid->initialize<AT::velocityY>(i, velY);
    }
    m_grid->pushCachToDevice();

    // swap buffers and ready for rendering
    m_grid->swapAndRender();

    // reset simulation state
    m_totalSimulatedTime = 0.0f;
    m_firstTimestep = true;
}

std::unique_ptr<Simulation> CosineAdvection::clone() const
{
    return std::make_unique<CosineAdvection>(*this);
}

void CosineAdvection::simulateOnce()
{
    if(m_cs->getType() != CSType::geographical2d)
        return;

    simulateOnceImpl(static_cast<GeographicalCoordinates2D&>( *(this->m_cs)));
}

__global__ void poleAdvectionA(ShallowWaterGrid::ReferenceType grid, GeographicalCoordinates2D coordinateSystem,
                               mpu::VectorReference<float> phiPlusK, mpu::VectorReference<float> vortPlusCor,
                               float timestep, bool useLeapfrog, float diffusion, float corOrAngvel)
{
    GeographicalCoordinates2D cs = coordinateSystem;

    // updates geopotential for all non boundary cells
    // also calculates kinetic energy per unit mass
    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-cs.hasBoundary().y ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);
            float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

            // read values of quantities
            const float phi = grid.read<AT::geopotential>(cellId);
            const float velRightX = grid.read<AT::velocityX>(cellId);
            const float velForY   = grid.read<AT::velocityY>(cellId);
            const float velLeftX  = grid.read<AT::velocityX>(cs.getLeftNeighbor(cellId));
            const float velBackY  = grid.read<AT::velocityY>(cs.getBackwardNeighbor(cellId));
            const float velForX  = grid.read<AT::velocityX>(cs.getForwardNeighbor(cellId)); // used for vorticity
            const float velRightY  = grid.read<AT::velocityY>(cs.getRightNeighbor(cellId)); // used for vorticity

            // compute kinetic energy per unit mass
            const float velX = (velLeftX + velRightX) * 0.5f;
            const float velY = (velForY + velBackY) * 0.5f;
            const float kinEnergy = (velX * velX + velY * velY) * 0.5f;
            phiPlusK[cellId] = kinEnergy + phi;

            // calculate vorticity and coriolis parameter
            // if this looks strange consider where values are located on the C grid
            const float2 vortPos = cellPos + 0.5f * make_float2(cs.getCellSize()); // position where vorticity is computed
            const float vort = curl2d(velForY, velRightY, velRightX, velForX, vortPos, cs);
            float cor;
            if(cs.getType() == CSType::geographical2d)
                cor = 2*corOrAngvel*sin(vortPos.y);
            else if(cs.getType() == CSType::cartesian2d)
                cor = corOrAngvel;
            else
                cor = 0.0f;
            vortPlusCor[cellId] = vort + cor;

            // write potential vorticity
            grid.write<AT::potentialVort>(cellId, abs(vort+cor) / phi);

            // compute geopotential advection time derivative dPhi/dt
            const float divv = divergence2d( velLeftX, velRightX, velBackY, velForY, cellPos, cs);
            float dphi_dt = -divv * phi;

            if(diffusion > 0)
            {
                const float phiLeft = grid.read<AT::geopotential>(cs.getLeftNeighbor(cellId));
                const float phiRight = grid.read<AT::geopotential>(cs.getRightNeighbor(cellId));
                const float phiFor = grid.read<AT::geopotential>(cs.getForwardNeighbor(cellId));
                const float phiBack = grid.read<AT::geopotential>(cs.getBackwardNeighbor(cellId));

                // compute geopotential diffusion
                const float lapphi = laplace2d(phiLeft,phiRight,phiBack,phiFor,phi,cellPos,cs);
                dphi_dt += diffusion * lapphi;
            }

            // compute values at t+1
            float nextPhi;
            if(useLeapfrog)
            {
                const float prevPhi = grid.readPrev<AT::geopotential>(cellId);
                nextPhi = prevPhi + dphi_dt * 2.0f*timestep;
            }
            else
                nextPhi = phi + dphi_dt * timestep;

            grid.write<AT::geopotential>(cellId,nextPhi);
        }
}

__global__ void poleAdvectionB(ShallowWaterGrid::ReferenceType grid, GeographicalCoordinates2D coordinateSystem,
                               mpu::VectorReference<const float> phiPlusK, mpu::VectorReference<float> vortPlusCor,
                               float timestep, bool useLeapfrog)
{
    GeographicalCoordinates2D cs = coordinateSystem;

    // updates all non boundary velocities
    // TODO: handle velocities parallel to the boundary
    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-2*cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-2*cs.hasBoundary().y ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);
            float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

            // read values of quantities
            const float phiKRight = phiPlusK[cs.getRightNeighbor(cellId)];
            const float phiKForward = phiPlusK[cs.getForwardNeighbor(cellId)];
            const float phiK = phiPlusK[cellId];
            const float vortCorLeft = vortPlusCor[cs.getLeftNeighbor(cellId)];
            const float vortCorBack = vortPlusCor[cs.getBackwardNeighbor(cellId)];
            const float vortCor = vortPlusCor[cellId];
            const float velX = grid.read<AT::velocityX>(cellId);
            const float velY = grid.read<AT::velocityY>(cellId);

            // compute dvX/dt and dvY/dt
            const float2 gradPhiK = gradient2d(phiK,phiKRight,phiK,phiKForward,cellPos,cs);
            const float dvX_dt = (vortCor+vortCorBack)*0.5f*velY -gradPhiK.x;
            const float dvY_dt = -(vortCor+vortCorLeft)*0.5f*velX -gradPhiK.y;

            // compute values at t+1
            float nextVelX;
            float nextVelY;
            if(useLeapfrog)
            {
                const float prevVelX = grid.readPrev<AT::velocityX>(cellId);
                const float prevVelY = grid.readPrev<AT::velocityY>(cellId);

                nextVelX = prevVelX + dvX_dt * 2.0f*timestep;
                nextVelY = prevVelY + dvY_dt * 2.0f*timestep;
            }
            else
            {
                nextVelX = velX + dvX_dt * timestep;
                nextVelY = velY + dvY_dt * timestep;
            }
            grid.write<AT::velocityX>(cellId,nextVelX);
            grid.write<AT::velocityY>(cellId,nextVelY);
        }
}


void CosineAdvection::simulateOnceImpl(GeographicalCoordinates2D& cs)
{
    dim3 blocksize{16,16,1};
    dim3 numBlocks{ static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().x ,blocksize.x)),
                    static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().y ,blocksize.y)), 1};

    poleAdvectionA << < numBlocks, blocksize >> > (m_grid->getGridReference(),cs,m_phiPlusKBuffer.getVectorReference(),
            m_vortPlusCor.getVectorReference(), m_timestep, !m_firstTimestep && m_useLeapfrog,
            m_geopotDiffusion, m_angularVelocitySI);
    poleAdvectionB << < numBlocks, blocksize >> > (m_grid->getGridReference(),cs,m_phiPlusKBuffer.getVectorReference(),
            m_vortPlusCor.getVectorReference(), m_timestep, !m_firstTimestep && m_useLeapfrog);

    m_totalSimulatedTime += m_timestep;
    m_firstTimestep = false;
}

GridBase& CosineAdvection::getGrid()
{
    return *m_grid;
}

std::string CosineAdvection::getDisplayName()
{
    return "Shallow Water Model";
}