#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * ShallowWaterModel.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the ShallowWaterModel class
 *
 * Copyright (c) 2020 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "CosineAdvection.h"

#include <mpUtils/mpUtils.h>
#include <mpUtils/mpGraphics.h>
#include <mpUtils/mpCuda.h>

#include "../GridReference.h"
#include "../finiteDifferences.h"
#include "../boundaryConditions.h"
//--------------------

// function definitions of the ShallowWaterModel class
//-------------------------------------------------------------------

void CosineAdvection::showCreationOptions()
{
    ImGui::Text("Test Case number 1 from David L. Williamson 1992.");
    ImGui::DragFloat("Wind Angle offset (alpha) in rad", &m_alpha, 0.001f,0.0,M_PI_2);
    ImGui::DragFloat("Wind Velocity (u0) in m/s", &m_u0SI, 0.001f);
    ImGui::DragFloat("Earth radius (a) in m", &m_earthRadiusSI);
    ImGui::DragFloat("Angular Velocity in rad/m", &m_angularVelocitySI, 0.00001f, 0.00001f, 5.0f, "%.8f");
    ImGui::DragFloat2("position of cosine bell", &m_cosineBellCenter.x, 0.001);
    ImGui::DragFloat("cosine bell radius (R) in m", &m_cosineBellRadiusSI, 1.0f);
    ImGui::DragFloat("Internal time unit in s", &m_timeUnit, 0.1f, 1.0);
}

void CosineAdvection::showBoundaryOptions(const CoordinateSystem& cs)
{
}

void CosineAdvection::showSimulationOptions()
{
    if(ImGui::DragFloat("Angular Velocity", &m_angularVelocitySI, 0.00001f, 0.00001, 5.0f, "%.5f"))
    {
        m_angularVelocity = m_angularVelocitySI * m_timeUnit;
    }

    ImGui::Checkbox("Use Leapfrog",&m_useLeapfrog);
    ImGui::DragFloat("Timestep",&m_timestep,0.000001,0.000001f,1.0,"%.6f");
    ImGui::Text("Simulated Time units: %f", m_totalSimulatedTime);
}

std::shared_ptr<GridBase> CosineAdvection::recreate(std::shared_ptr<CoordinateSystem> cs)
{
    m_cs = cs;
    m_grid = std::make_shared<ShallowWaterGrid>(m_cs->getNumGridCells());

    if(m_cs->getType() != CSType::geographical2d)
    {
        logERROR("CosineAdvection") << "Pole advection test loaded in cartesian cordinates!";
        tinyfd_messageBox("Error","Advection Test only works for geographical coordinates",
                          "ok", "error",1);
        return m_grid;
    }

    // scale units
    m_lengthUnit = m_earthRadiusSI / m_cs->getMinCoord().z;
    logINFO("PoleAdvectionTest") << "Internal length unit: " << m_lengthUnit << " meter";
    logINFO("PoleAdvectionTest") << "Internal time unit: " << m_timeUnit << " seconds";

    m_earthRadius = m_earthRadiusSI / m_lengthUnit;
    m_u0 = m_u0SI / m_lengthUnit * m_timeUnit;
    m_angularVelocity = m_angularVelocitySI * m_timeUnit;
    m_cosineBellRadius = m_cosineBellRadiusSI / m_lengthUnit;
    m_h0 = m_h0SI / m_lengthUnit;
    m_g = m_gSI / m_lengthUnit * m_timeUnit * m_timeUnit;

    logINFO("PoleAdvectionTest") << "Settings in internal units: earth radius: " << m_earthRadius << ", u0: " << m_u0
                                 << ", angular velocity: " << m_angularVelocity
                                 << ", cosine bell radius: " << m_cosineBellRadius
                                 << ", cosine bell heigt: " << m_h0
                                 << ", g: " << m_g
                                 ;

    reset();
    return m_grid;
}

void CosineAdvection::reset()
{
    m_grid->cacheOverwrite();

    float cosAlpha = cos(m_alpha);
    float sinAlpha = sin(m_alpha);

    float sinLatCenter = sin(m_cosineBellCenter.y);
    float cosLatCenter = cos(m_cosineBellCenter.y);

    // create initial conditions using gaussian
    for(int i : mpu::Range<int>(m_grid->size()))
    {
        float3 cp = m_cs->getCellCoordinate(i);
        float3 cv = m_cs->getCellCoordinate(i) + m_cs->getCellSize()*0.5f;

        float velX = m_u0*(cos(cv.y)*cosAlpha + sin(cv.y)*cos(cv.x)*sinAlpha);
        float velY = -m_u0*sin(cv.y)*sinAlpha;

        float geopotential = 0;
        float r = m_earthRadius * acos( sinLatCenter*sin(cp.y) + cosLatCenter*cos(cp.y)*cos(cp.x - m_cosineBellCenter.x));
        if(r < m_cosineBellRadius)
        {
            float h = (m_h0/2.0f) * (1.0f + cos( M_PI * r / m_cosineBellRadius ));
            geopotential = m_g*h;
        }

        m_grid->initialize<AT::geopotential>(i, geopotential);
        m_grid->initialize<AT::velocityX>(i, velX);
        m_grid->initialize<AT::velocityY>(i, velY);
    }
    m_grid->pushCachToDevice();

    // swap buffers and ready for rendering
    m_grid->swapAndRender();

    // reset simulation state
    m_totalSimulatedTime = 0.0f;
    m_firstTimestep = true;
}

std::unique_ptr<Simulation> CosineAdvection::clone() const
{
    return std::make_unique<CosineAdvection>(*this);
}

void CosineAdvection::simulateOnce()
{
    if(m_cs->getType() != CSType::geographical2d)
        return;

    simulateOnceImpl(static_cast<GeographicalCoordinates2D&>( *(this->m_cs)));
}

__global__ void poleAdvectionA(ShallowWaterGrid::ReferenceType grid, GeographicalCoordinates2D coordinateSystem,
                               float timestep, bool useLeapfrog, float angularVelocity)
{
    GeographicalCoordinates2D cs = coordinateSystem;

    // updates geopotential for all non boundary cells
    // also calculates kinetic energy per unit mass
    for(int x : mpu::gridStrideRange( cs.hasBoundary().x, cs.getNumGridCells3d().x-cs.hasBoundary().x ))
        for(int y : mpu::gridStrideRangeY( cs.hasBoundary().y, cs.getNumGridCells3d().y-cs.hasBoundary().y ))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);
            float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

            // read values of quantities
            const float phi = grid.read<AT::geopotential>(cellId);
            const float phiLeft = grid.read<AT::geopotential>(cs.getLeftNeighbor(cellId));
            const float phiRight = grid.read<AT::geopotential>(cs.getRightNeighbor(cellId));
            const float phiBack = grid.read<AT::geopotential>(cs.getBackwardNeighbor(cellId));
            const float phiFor = grid.read<AT::geopotential>(cs.getForwardNeighbor(cellId));

            const float velLeftX  = grid.read<AT::velocityX>(cs.getLeftNeighbor(cellId));
            const float velRightX = grid.read<AT::velocityX>(cellId);
            const float velBackY  = grid.read<AT::velocityY>(cs.getBackwardNeighbor(cellId));
            const float velForY   = grid.read<AT::velocityY>(cellId);
//            const float velForX  = grid.read<AT::velocityX>(cs.getForwardNeighbor(cellId)); // used for vorticity
//            const float velRightY  = grid.read<AT::velocityY>(cs.getRightNeighbor(cellId)); // used for vorticity

            // calculate vorticity and coriolis parameter
            // if this looks strange consider where values are located on the C grid
//            const float2 vortPos = cellPos + 0.5f * make_float2(cs.getCellSize()); // position where vorticity is computed
//            const float vort = curl2d(velForY, velRightY, velRightX, velForX, vortPos, cs);
//            float cor;
//            if(cs.getType() == CSType::geographical2d)
//                cor = 2*corOrAngvel*sin(vortPos.y);
//            else if(cs.getType() == CSType::cartesian2d)
//                cor = corOrAngvel;
//            else
//                cor = 0.0f;
//            vortPlusCor[cellId] = vort + cor;
//
//            // write potential vorticity
//            grid.write<AT::potentialVort>(cellId, abs(vort+cor) / phi);

            // compute geopotential advection time derivative dPhi/dt
            const float phiHalfLeft = 0.5f*(phi+phiLeft);
            const float phiHalfRight = 0.5f*(phi+phiRight);
            const float phiHalfBack = 0.5f*(phi+phiBack);
            const float phiHalfFor = 0.5f*(phi+phiFor);
            float dphi_dt = -divergence2d( velLeftX*phiHalfLeft, velRightX*phiHalfRight, velBackY*phiHalfBack, velForY*phiHalfFor, cellPos, cs);

            // compute values at t+1
            float nextPhi;
            if(useLeapfrog)
            {
                const float prevPhi = grid.readPrev<AT::geopotential>(cellId);
                nextPhi = prevPhi + dphi_dt * 2.0f*timestep;
            }
            else
                nextPhi = phi + dphi_dt * timestep;

            grid.write<AT::potentialVort>(cellId, dphi_dt);
            grid.write<AT::geopotential>(cellId, nextPhi);
        }
}

void CosineAdvection::simulateOnceImpl(GeographicalCoordinates2D& cs)
{
    dim3 blocksize{16,16,1};
    dim3 numBlocks{ static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().x ,blocksize.x)),
                    static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().y ,blocksize.y)), 1};

    poleAdvectionA<<< numBlocks, blocksize >>>(m_grid->getGridReference(),cs,m_timestep, !m_firstTimestep && m_useLeapfrog, m_angularVelocity);

    m_totalSimulatedTime += m_timestep;
    m_firstTimestep = false;
}

GridBase& CosineAdvection::getGrid()
{
    return *m_grid;
}

std::string CosineAdvection::getDisplayName()
{
    return "Shallow Water Model";
}