#include "hip/hip_runtime.h"
/*
 * CIRCULATION
 * ShallowWaterModel.cpp
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Implements the ShallowWaterModel class
 *
 * Copyright (c) 2020 Hendrik Schwanekamp
 *
 */

// includes
//--------------------
#include "CosineAdvection.h"

#include <mpUtils/mpUtils.h>
#include <mpUtils/mpGraphics.h>
#include <mpUtils/mpCuda.h>

#include "../GridReference.h"
#include "../finiteDifferences.h"
#include "../boundaryConditions.h"
#include "../interpolation.h"
//--------------------

// function definitions of the ShallowWaterModel class
//-------------------------------------------------------------------

void CosineAdvection::showCreationOptions()
{
    ImGui::Text("Test Case number 1 from David L. Williamson 1992.");
    ImGui::DragFloat("Earth radius (a) in m", &m_earthRadiusSI);
    ImGui::DragFloat("Angular Velocity in rad/m", &m_angularVelocitySI, 0.00001f, 0.00001f, 5.0f, "%.8f");
    ImGui::DragFloat("Internal time unit in s", &m_timeUnit, 0.1f, 1.0);
    ImGui::DragFloat("Wind Angle offset (alpha) in rad", &m_alpha, 0.001f,0.0,M_PI_2);
    ImGui::DragFloat("Wind Velocity (u0) in m/s", &m_u0SI, 0.001f);
    ImGui::DragFloat2("position of cosine bell", &m_cosineBellCenter.x, 0.001);
    ImGui::DragFloat("cosine bell radius (R) in m", &m_cosineBellRadiusSI, 1.0f);
}

void CosineAdvection::showBoundaryOptions(const CoordinateSystem& cs)
{
}

void CosineAdvection::showSimulationOptions()
{
    if(ImGui::DragFloat("Wind Angle offset (alpha) in rad", &m_alpha, 0.001f,0.0,M_PI_2))
        buildWindField();

    if(ImGui::DragFloat("Wind Velocity (u0) in m/s", &m_u0SI, 0.001f))
        buildWindField();

    if(ImGui::DragFloat("Angular Velocity", &m_angularVelocitySI, 0.00001f, 0.00001, 5.0f, "%.5f"))
    {
        m_angularVelocity = m_angularVelocitySI * m_timeUnit;
    }

    ImGui::Checkbox("Use Leapfrog",&m_useLeapfrog);
    ImGui::DragFloat("Timestep",&m_timestep,0.000001,0.000001f,1.0,"%.6f");
    ImGui::Text("Simulated Time units: %f", m_totalSimulatedTime);
}

std::shared_ptr<GridBase> CosineAdvection::recreate(std::shared_ptr<CoordinateSystem> cs)
{
    m_cs = cs;
    m_grid = std::make_shared<ShallowWaterGrid>(m_cs->getNumGridCells());

    if(m_cs->getType() != CSType::geographical2d)
    {
        logERROR("CosineAdvection") << "Pole advection test loaded in cartesian cordinates!";
        tinyfd_messageBox("Error","Advection Test only works for geographical coordinates",
                          "ok", "error",1);
        return m_grid;
    }

    // scale units
    m_lengthUnit = m_earthRadiusSI / m_cs->getMinCoord().z;
    logINFO("PoleAdvectionTest") << "Internal length unit: " << m_lengthUnit << " meter";
    logINFO("PoleAdvectionTest") << "Internal time unit: " << m_timeUnit << " seconds";

    m_earthRadius = m_earthRadiusSI / m_lengthUnit;
    m_u0 = m_u0SI / m_lengthUnit * m_timeUnit;
    m_angularVelocity = m_angularVelocitySI * m_timeUnit;
    m_cosineBellRadius = m_cosineBellRadiusSI / m_lengthUnit;
    m_h0 = m_h0SI / m_lengthUnit;
    m_g = m_gSI / m_lengthUnit * m_timeUnit * m_timeUnit;

    logINFO("PoleAdvectionTest") << "Settings in internal units: earth radius: " << m_earthRadius << ", u0: " << m_u0
                                 << ", angular velocity: " << m_angularVelocity
                                 << ", cosine bell radius: " << m_cosineBellRadius
                                 << ", cosine bell heigt: " << m_h0
                                 << ", g: " << m_g
                                 ;
    buildWindField();
    reset();
    return m_grid;
}

void CosineAdvection::buildWindField()
{
    m_grid->cacheOnHost();

    float cosAlpha = cos(m_alpha);
    float sinAlpha = sin(m_alpha);

    float sinLatCenter = sin(m_cosineBellCenter.y);
    float cosLatCenter = cos(m_cosineBellCenter.y);

    // create solid body rotation velocity field
    for(int i : mpu::Range<int>(m_grid->size()))
    {
        float3 cv = m_cs->getCellCoordinate(i) + m_cs->getCellSize()*0.5f;

        float velX = m_u0*( cos(cv.y)*cosAlpha + sin(cv.y)*cos(cv.x)*sinAlpha);
        float velY = -m_u0*sin(cv.x)*sinAlpha;

        m_grid->initialize<AT::velocityX>(i, velX);
        m_grid->initialize<AT::velocityY>(i, velY);
    }
    m_grid->pushCachToDevice();

    // swap buffers and ready for rendering
    m_grid->swapAndRender();
}

void CosineAdvection::reset()
{
    m_grid->cacheOnHost();

    std::default_random_engine rng(mpu::getRanndomSeed());
    std::uniform_real_distribution<float> dist(0.9,1.1);

    float cosAlpha = cos(m_alpha);
    float sinAlpha = sin(m_alpha);

    float sinLatCenter = sin(m_cosineBellCenter.y);
    float cosLatCenter = cos(m_cosineBellCenter.y);

    // create initial conditions using cosine bell
    for(int i : mpu::Range<int>(m_grid->size()))
    {
        float3 cp = m_cs->getCellCoordinate(i);
        float geopotential = 0;
        float r = m_earthRadius * acos( sinLatCenter*sin(cp.y) + cosLatCenter*cos(cp.y)*cos(cp.x - m_cosineBellCenter.x));
        if(r < m_cosineBellRadius)
        {
            float h = dist(rng) * (m_h0/2.0f) * (1.0f + cos( M_PI * r / m_cosineBellRadius ));
            geopotential = m_g*h;
        }

        m_grid->initialize<AT::geopotential>(i, geopotential);
    }
    m_grid->pushCachToDevice();

    // swap buffers and ready for rendering
    m_grid->swapAndRender();

    // reset simulation state
    m_totalSimulatedTime = 0.0f;
    m_firstTimestep = true;
}

std::unique_ptr<Simulation> CosineAdvection::clone() const
{
    return std::make_unique<CosineAdvection>(*this);
}

void CosineAdvection::simulateOnce()
{
    if(m_cs->getType() != CSType::geographical2d)
        return;

    simulateOnceImpl(static_cast<GeographicalCoordinates2D&>( *(this->m_cs)));
}

__global__ void poleAdvectionA(ShallowWaterGrid::ReferenceType grid, GeographicalCoordinates2D coordinateSystem,
                               float timestep, bool useLeapfrog, float angularVelocity)
{
    GeographicalCoordinates2D cs = coordinateSystem;

    // updates geopotential for all non boundary cells
    // also calculates kinetic energy per unit mass
    for(int x : mpu::gridStrideRange( 0, cs.getNumGridCells3d().x))
        for(int y : mpu::gridStrideRangeY( 0, cs.getNumGridCells3d().y))
        {
            int3 cell{x,y,0};
            int cellId = cs.getCellId(cell);
            float2 cellPos = make_float2( cs.getCellCoordinate3d(cell) );

            // read values of quantities
            const float phi = grid.read<AT::geopotential>(cellId);
            const float phiLeft = grid.read<AT::geopotential>(cs.getLeftNeighbor(cellId));
            const float phiRight = grid.read<AT::geopotential>(cs.getRightNeighbor(cellId));

            const float velLeftX  = grid.read<AT::velocityX>(cs.getLeftNeighbor(cellId));
            const float velRightX = grid.read<AT::velocityX>(cellId);
            float velBackY;
            float velForY;

            // compute phi half, meaning phi half way between cells
            const float phiHalfLeft = 0.5f*(phi+phiLeft);
            const float phiHalfRight = 0.5f*(phi+phiRight);
            float phiHalfBack;
            float phiHalfFor;

            // handle lower boundary
            if(cell.y == 0)
            {
                velBackY = 0;
                phiHalfBack = 0;
                continue; // we cant handle this case right now
            } else {
                float phiBack = grid.read<AT::geopotential>(cs.getBackwardNeighbor(cellId));
                velBackY  = grid.read<AT::velocityY>(cs.getBackwardNeighbor(cellId));
                phiHalfBack = 0.5f*(phi+phiBack);
            }

//            bool print = cell.x == 256 || cell.x == 0;
//            bool print = cell.x == 384 || cell.x == 128;
//            bool print = cell.x == 127 || cell.x == 128 || cell.x == 129 || cell.x == 384;
            // handle upper boundary
            if(cell.y == cs.getNumGridCells3d().y-1)
            {
//                continue; // we cant handle this case right now

                // find where we need phiHalfFor in extended coordinates
                float2 neighborPosExt = cellPos;
                neighborPosExt.y += cs.getCellSize().y*0.5f;

                // get the same in position in normal coordinates by wrapping around the pole
                float2 neighborPos = neighborPosExt;
                if(neighborPos.y > M_PIf32*0.5)
                {
                    neighborPos.y = M_PIf32 - neighborPos.y;
                    neighborPos.x = fmod(neighborPos.x+M_PIf32,2*M_PIf32);
                }

//                if(print) printf("cell %i pos: (%f,%f), nbPosExt: (%f,%f), nbPos: (%f,%f)\n",cell.x, cellPos.x, cellPos.y,
//                        neighborPosExt.x, neighborPosExt.y, neighborPos.x, neighborPos.y);

                // if we are in the polar region (outside of the coordinate bounds) special care must be taken
                if( neighborPos.y > cs.getMaxCoord().y)
                {
//                    if(print) printf("cell %i geopotential using interpolatePole\n",cell.x);
                    phiHalfFor = interpolateNorthPole2D<AT::geopotential>(cellPos,phi,neighborPosExt.y,grid,cs,float2{0,0},false);
//                    if(print && phiHalfFor < 0)
//                        printf("cell %i has negative neighbor\n",cell.x);
                } else
                {
//                    if(print) printf("cell %i geopotential using interpolate2d\n",cell.x);
                    phiHalfFor = readInterpolated2D<AT::geopotential>(neighborPos,grid,cs);
                }

                // velocity X has an offset on the c grid, but we need to load it at the same position as phi
                float2 offset{0,cs.getCellSize().y*0.5f};
                if( neighborPos.y > cs.getMaxCoord().y-offset.y)
                {
//                    if(print) printf("cell %i velocity using interpolatePole\n",cell.x);
                    velForY = interpolateNorthPole2D<AT::velocityY>(cellPos-offset, velBackY,neighborPosExt.y,grid,cs,offset,true);
                } else
                {
//                    if(print) printf("cell %i velocity using interpolate2D\n",cell.x);
                    velForY = -readInterpolated2D<AT::velocityY>(neighborPos,grid,cs,offset);
                }

//                if(print) printf("cell %i geo: %f geoBack: %f, geoFor %f\n",cell.x,phi,phiHalfBack,phiHalfFor);
//                if(print) printf("cell %i velBack: %f, velFor %f\n",cell.x,velBackY,velForY);

            } else {
                float phiFor = grid.read<AT::geopotential>(cs.getForwardNeighbor(cellId));
                velForY = grid.read<AT::velocityY>(cellId);
                phiHalfFor = 0.5f*(phi+phiFor);
            }

//            const float velForX  = grid.read<AT::velocityX>(cs.getForwardNeighbor(cellId)); // used for vorticity
//            const float velRightY  = grid.read<AT::velocityY>(cs.getRightNeighbor(cellId)); // used for vorticity

            // calculate vorticity and coriolis parameter
            // if this looks strange consider where values are located on the C grid
//            const float2 vortPos = cellPos + 0.5f * make_float2(cs.getCellSize()); // position where vorticity is computed
//            const float vort = curl2d(velForY, velRightY, velRightX, velForX, vortPos, cs);
//            float cor;
//            if(cs.getType() == CSType::geographical2d)
//                cor = 2*corOrAngvel*sin(vortPos.y);
//            else if(cs.getType() == CSType::cartesian2d)
//                cor = corOrAngvel;
//            else
//                cor = 0.0f;
//            vortPlusCor[cellId] = vort + cor;
//
//            // write potential vorticity
//            grid.write<AT::potentialVort>(cellId, abs(vort+cor) / phi);

            // compute geopotential advection time derivative dPhi/dt
            float dphi_dt = -divergence2d( velLeftX*phiHalfLeft, velRightX*phiHalfRight, velBackY*phiHalfBack, velForY*phiHalfFor, cellPos, cs);

            // compute values at t+1
            float nextPhi;
            if(useLeapfrog)
            {
                const float prevPhi = grid.readPrev<AT::geopotential>(cellId);
                nextPhi = prevPhi + dphi_dt * 2.0f*timestep;
            }
            else
                nextPhi = phi + dphi_dt * timestep;

            grid.write<AT::potentialVort>(cellId, dphi_dt);
            grid.write<AT::geopotential>(cellId, nextPhi);
        }
}

void CosineAdvection::simulateOnceImpl(GeographicalCoordinates2D& cs)
{
    dim3 blocksize{16,16,1};
    dim3 numBlocks{ static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().x ,blocksize.x)),
                    static_cast<unsigned int>(mpu::numBlocks( cs.getNumGridCells3d().y ,blocksize.y)), 1};

    poleAdvectionA<<< numBlocks, blocksize >>>(m_grid->getGridReference(),cs,m_timestep, !m_firstTimestep && m_useLeapfrog, m_angularVelocity);

    m_totalSimulatedTime += m_timestep;
    m_firstTimestep = false;
}

GridBase& CosineAdvection::getGrid()
{
    return *m_grid;
}

std::string CosineAdvection::getDisplayName()
{
    return "Shallow Water Model";
}